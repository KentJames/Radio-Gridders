//C++ Includes
#include <iostream>
#include <fstream>
#include <cstdlib>

//CUDA Includes
#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>
#include "hip/hip_runtime.h"
#include "math.h"
#include "hip/hip_runtime_api.h"

//Our Include
#include "wtowers_common.h"

#define cudaError_check(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


//W-Towers Wrapper.
__host__ hipError_t wtowers_host(const char* visfile, const char* wkernfile, int grid_size,
			   double theta,  double lambda, double bl_min, double bl_max,
				  int iter){

  hipError_t error;
  hipEvent_t start, stop;
  float elapsedTime;

  struct vis_data *vis_dat;
  struct w_kernel_data *wkern_dat;


  cudaError_check(hipMallocManaged((void **)&vis_dat, sizeof(struct vis_data), hipMemAttachGlobal));
  cudaError_check(hipMallocManaged((void **)&wkern_dat, sizeof(struct w_kernel_data), hipMemAttachGlobal));

  int error_hdf5;
  error_hdf5 = load_vis_CUDA(visfile,vis_dat,bl_min,bl_max);
  if (error_hdf5) {
    std::cout << "Failed to Load Visibilities \n";
    return error;
  }
  error_hdf5 = load_wkern_CUDA(wkernfile, theta, wkern_dat);
  if (error_hdf5) {
    std::cout << "Failed to Load W-Kernels \n";
    return error;
  }

  int total_gs = grid_size * grid_size;
  
  hipDoubleComplex *grid_dev, *grid_host;

  cudaError_check(hipMalloc((void **)&grid_dev, total_gs * sizeof(hipDoubleComplex)));
  cudaError_check(hipHostMalloc((void **)&grid_host, total_gs * sizeof(hipDoubleComplex)));
  

  
  return error;

}
