#include "hip/hip_runtime.h"
//C++ Includes
#include <iostream>
#include <fstream>
#include <cstdlib>
#include <cassert>

//CUDA Includes
#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>
#include "hip/hip_runtime.h"
#include "math.h"
#include "hip/hip_runtime_api.h"

//Our Include
#include "wtowers_common.h"



/*****************************
      CUDA Error Checker
******************************/
 
#define cudaError_check(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


/*****************************
        Device Functions
 *****************************/


__host__ __device__ inline hipDoubleComplex cu_cexp_d (hipDoubleComplex z){

  hipDoubleComplex res;
  double t = exp (z.x);
  sincos (z.y, &res.y, &res.x);
  res.x *= t;
  res.y *= t;
  return res;

}


  
  

/******************************
            Kernels
*******************************/



//This is our Romein-style scatter gridder.
__global__ void scatter_grid_kernel(struct vis_data *vis, struct w_kernel_data *wkern,
				  hipDoubleComplex *uvgrid, int max_support, int grid_size){



}


__global__ void fresnel_pattern_kernel(hipDoubleComplex *subimg, hipDoubleComplex *subgrid,
				       hipDoubleComplex *fresnel, int subgrid_size, int w_plane){



}
				       






/******************************
	  Host Functions
*******************************/
 
__host__ inline void fft_shift(hipDoubleComplex *uvgrid, int grid_size) {

    // Shift the FFT
    assert(grid_size % 2 == 0);
    int x, y;
    for (y = 0; y < grid_size; y++) {
        for (x = 0; x < grid_size/2; x++) {
            int ix0 = y * grid_size + x;
            int ix1 = (ix0 + (grid_size+1) * (grid_size/2)) % (grid_size*grid_size);
            hipDoubleComplex temp = uvgrid[ix0];
            uvgrid[ix0] = uvgrid[ix1];
            uvgrid[ix1] = temp;
        }
    }

}



//W-Towers Wrapper.
__host__ hipError_t wtowers_CUDA(const char* visfile, const char* wkernfile, int grid_size,
			   double theta,  double lambda, double bl_min, double bl_max,
				  int subgrid_size, int subgrid_margin, double wincrement){
  //For Benchmarking.
  
  hipError_t error;
  hipEvent_t start, stop;
  float elapsedTime;

  // Load visibility and w-kernel data from HDF5 files.
  
  struct vis_data *vis_dat;
  struct w_kernel_data *wkern_dat;

  cudaError_check(hipMallocManaged((void **)&vis_dat, sizeof(struct vis_data), hipMemAttachGlobal));
  cudaError_check(hipMallocManaged((void **)&wkern_dat, sizeof(struct w_kernel_data), hipMemAttachGlobal));

  int error_hdf5;
  error_hdf5 = load_vis_CUDA(visfile,vis_dat,bl_min,bl_max);
  if (error_hdf5) {
    std::cout << "Failed to Load Visibilities \n";
    return error;
  }
  error_hdf5 = load_wkern_CUDA(wkernfile, theta, wkern_dat);
  if (error_hdf5) {
    std::cout << "Failed to Load W-Kernels \n";
    return error;
  }


  //Allocate our main grid.
  
  int total_gs = grid_size * grid_size;
  
  hipDoubleComplex *grid_dev, *grid_host;
  cudaError_check(hipMalloc((void **)&grid_dev, total_gs * sizeof(hipDoubleComplex)));
  cudaError_check(hipHostMalloc((void **)&grid_host, total_gs * sizeof(hipDoubleComplex)));

  int subgrid_mem_size = sizeof(hipDoubleComplex) * subgrid_size * subgrid_size;


  //Create the fresnel interference pattern for the W-Dimension
  //See Tim Cornwells paper on W-Projection for more information.
  
  hipDoubleComplex *wtransfer;
  cudaError_check(hipMallocManaged((void **)&wtransfer, subgrid_mem_size, hipMemAttachGlobal));

  int x,y;
  for (y=0; y < subgrid_size; ++y){

    for (x=0; x < subgrid_size; ++x){

      double l = theta * (double)(x - subgrid_size / 2) / subgrid_size;
      double m = theta * (double)(y - subgrid_size / 2) / subgrid_size;
      double ph = wincrement * (1 - sqrt(1 - l*l - m*m));

      hipDoubleComplex wtrans = make_hipDoubleComplex(0, 2 * M_PI * ph);
      wtransfer[y * subgrid_size + x] = cu_cexp_d(wtrans);
    }

  }

  //Create FFT Plans for our frequent fft's.

  hipfftHandle fft_plan;
  hipfftPlan2d(&fft_plan,subgrid_size,subgrid_size,HIPFFT_D2Z);


  //Allocate subgrids/subimgs on the GPU
  
  assert( grid_size % subgrid_size == 0);
  int chunk_count_1d = grid_size / subgrid_size;
  int total_chunks = chunk_count_1d * chunk_count_1d;

  hipDoubleComplex **subgrids, **subimgs;

  cudaError_check(hipMallocManaged(&subgrids, total_chunks * sizeof(hipDoubleComplex)));
  cudaError_check(hipMallocManaged(&subimgs, total_chunks * sizeof(hipDoubleComplex)));

  //Create streams for each tower and allocate our chunks on GPU memory.
  
  hipStream_t streams[total_chunks];
  for(int i = 0; i < total_chunks; ++i){

    hipStreamCreate(&streams[i]);

    cudaError_check(hipMallocManaged(subgrids + i, subgrid_mem_size * sizeof(hipDoubleComplex)));
    cudaError_check(hipMallocManaged(subimgs + i, subgrid_mem_size * sizeof(hipDoubleComplex)));

  }

  

  
  return error;

}
