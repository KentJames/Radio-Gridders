#include "hip/hip_runtime.h"
//C++ Includes
#include <iostream>
#include <fstream>
#include <cstdlib>
#include <cassert>

//CUDA Includes
#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>
#include "hip/hip_runtime.h"
#include "math.h"
#include "hip/hip_runtime_api.h"

//Our Include
#include "wtowers_common.h"



/*****************************
      CUDA Error Checker
******************************/
 
#define cudaError_check(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


/*****************************
        Device Functions
 *****************************/

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600


#else //Pre-pascal devices.

__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

#endif

__host__ __device__ inline hipDoubleComplex cu_cexp_d (hipDoubleComplex z){

  hipDoubleComplex res;
  double t = exp (z.x);
  sincos (z.y, &res.y, &res.x);
  res.x *= t;
  res.y *= t;
  return res;

}

__host__ __device__ inline static double uvw_lambda(struct bl_data *bl_data,
				  int time, int freq, int uvw) {
    return bl_data->uvw[3*time+uvw] * bl_data->freq[freq] / c;
  }



__host__ __device__ inline static int2 getcoords_xy(double u, double v, int grid_size,
				    double theta, int max_support){

  int2 xy;

  xy.x = ((int)floor(theta * u + 0.5) + (grid_size/2)) % max_support;
  xy.y = ((int)floor(theta * v + 0.5) + (grid_size/2)) % max_support;

  return xy;
    

}


__host__ __device__ inline static void frac_coord(int grid_size, int kernel_size, int oversample,
                              double theta,
                              struct bl_data *bl_data,
                              int time, int freq,
                              double d_u, double d_v,
                              int *grid_offset, int *sub_offset) {
#ifdef ASSUME_UVW_0
    double x = 0, y = 0;
#else
    double x = theta * (uvw_lambda(bl_data, time, freq, 0) - d_u);
    double y = theta * (uvw_lambda(bl_data, time, freq, 1) - d_v);
#endif
    int flx = (int)floor(x + .5 / oversample);
    int fly = (int)floor(y + .5 / oversample);
    int xf = (int)floor((x - (double)flx) * oversample + .5);
    int yf = (int)floor((y - (double)fly) * oversample + .5);
    *grid_offset =
        (fly+grid_size/2-kernel_size/2)*grid_size +
        (flx+grid_size/2-kernel_size/2);
    *sub_offset = kernel_size * kernel_size * (yf*oversample + xf);
}


//From Kyrills implementation in SKA/RC
__device__ inline void scatter_grid_add(hipDoubleComplex *uvgrid, int grid_size, int grid_pitch,
					int grid_point_u, int grid_point_v, hipDoubleComplex sum){

  
  // Atomically add to grid. This is the bottleneck of this kernel.
  if (grid_point_u < 0 || grid_point_u >= grid_size ||
      grid_point_v < 0 || grid_point_v >= grid_size)
    return;

  // Bottom half? Mirror
  if (grid_point_u >= grid_size / 2) {
    grid_point_v = grid_size - grid_point_v - 1;
    grid_point_u = grid_size - grid_point_u - 1;
  }

  // Add to grid. This is the bottleneck of the entire kernel
  atomicAdd(&uvgrid[grid_point_u + grid_pitch*grid_point_v].x, sum.x);
  atomicAdd(&uvgrid[grid_point_u + grid_pitch*grid_point_v].y, sum.y);

}


//From Kyrills Implementation in SKA/RC. Modified to suit our data format.
//Assumes pre-binned (in u/v) data
__device__ inline void scatter_grid_point(
					  struct bl_data **bin, // Our bins of UV Data
					  int bl_count, // Number of baselines.
					  hipDoubleComplex *uvgrid, // Our main UV Grid
					  struct w_kernel_data *wkern, //Our W-Kernel
					  int max_supp, // Max size of W-Kernel
					  int myU, //Our assigned u/v points.
					  int myV, // ^^^
					  double wstep, // W-Increment 
					  int subgrid_size, //The size of our w-towers subgrid.
					  int subgrid_pitch, // Not too sure about ths one
					  int theta, // Field of View Size
					  int offset_u, // Offset from top left of main grid to t.l of subgrid.
					  int offset_v, // ^^^^
					  int offset_w
					  ){ 

  int grid_point_u = myU, grid_point_v = myV;
  hipDoubleComplex sum  = make_hipDoubleComplex(0.0,0.0);
  
  //  for (int i = 0; i < visibilities; i++) {
  int bl, time, freq;
  for (bl = 0; bl < bl_count; ++bl){
    struct bl_data *bl_d = bin[bl];
    for (time = 0; time < bl_d->time_count; ++time){
      for(freq = 0; freq < bl_d->freq_count; ++freq){
	// Load pre-calculated positions
	//int u = uvo[i].u, v = uvo[i].v;
	//	int u = (int)uvw_lambda(bl_d, time, freq, 0);
	//int v = (int)uvw_lambda(bl_d, time, freq, 1);
	double w = uvw_lambda(bl_d, time, freq, 2) - offset_w;
	int w_plane = fabs(w/wstep);

	//i
	int grid_offset, sub_offset;
	frac_coord(subgrid_size, wkern->size_x, wkern->oversampling,
		   theta, bl_d, time, freq, offset_u, offset_v, &grid_offset, &sub_offset);
	int u = floor(grid_offset / subgrid_size);
	int v = grid_offset % subgrid_size;

	// Determine convolution point. This is basically just an
	// optimised way to calculate
	//   myConvU = (myU - u) % max_supp
	//   myConvV = (myV - v) % max_supp
	//	int2 xy = getcoords_xy(u,v,subgrid_size,theta,max_supp);
	int myConvU = ((int)u - myU) % max_supp;
	int myConvV = ((int)v - myV) % max_supp;
	if (myConvU < 0) myConvU += max_supp;
	if (myConvV < 0) myConvV += max_supp;

	// Determine grid point. Because of the above we know here that
	//   myGridU % max_supp = myU
	//   myGridV % max_supp = myV
	int myGridU = u + myConvU
	  , myGridV = v + myConvV;

	// Grid point changed?
	if (myGridU != grid_point_u || myGridV != grid_point_v) {
	  // Atomically add to grid. This is the bottleneck of this kernel.
	  scatter_grid_add(uvgrid, subgrid_size, subgrid_pitch, grid_point_u, grid_point_v, sum);
	  // Switch to new point
	  sum = make_hipDoubleComplex(0.0, 0.0);
	  grid_point_u = myGridU;
	  grid_point_v = myGridV;
	}


	//TODO: Re-do the w-kernel/gcf for our data.

	short supp = short(wkern->size_x);	
	//	hipDoubleComplex px;
	hipDoubleComplex px = *(hipDoubleComplex*)&wkern->kern_by_w[w_plane].data[sub_offset + myConvU * supp + myConvV];
	//memcpy(&px, &pxc, sizeof(double __complex__));
	
	// Sum up
	hipDoubleComplex vi = *(hipDoubleComplex*)&bl_d->vis[time*bl_d->freq_count+freq];
	//memcpy(&vi,&visc,sizeof(double __complex__));
	
	if (grid_point_u >= subgrid_size / 2)
	  vi.y = -vi.y;
	sum = hipCfma(px, vi, sum);
      }
    }
  }

  // Add remaining sum to grid
  scatter_grid_add(uvgrid, subgrid_size, subgrid_pitch, grid_point_u, grid_point_v, sum);

}


/******************************
            Kernels
*******************************/

//This is our Romein-style scatter gridder.
__global__ void scatter_grid_kernel(struct bl_data **bin, // Baseline bin
				    int bl_count, // No. of baselines
				    struct vis_data *vis, // No. of visibilities
				    struct w_kernel_data *wkern, // No. of wkernels
				    hipDoubleComplex *uvgrid, //Our UV-Grid
				    int max_support, //  Convolution size
				    int subgrid_size, // Subgrid size
				    int subgrid_pitch, // Subgrid pitch (what is this?)
				    double wstep, // W-Increment
				    double theta, // Field of View
				    int offset_u, // Top left offset from top left main grid
				    int offset_v, // ^^^^
				    int offset_w // W Offset
				    ){
  
  for(int i = threadIdx.x; i < max_support * max_support; i += blockDim.x){

    int myU = i % max_support;
    int myV = floor((double)i / (double)max_support); // Double cast ensures nvcc uses CUDA floor.

    scatter_grid_point(bin, bl_count, uvgrid, wkern, max_support, myU, myV, wstep,
		       subgrid_size, subgrid_pitch, theta, offset_u, offset_v, offset_w);
		       
  }
  

}

//For multiplying the fresnel pattern.
__global__ void fresnel_pattern_kernel(hipDoubleComplex *subimg, hipDoubleComplex *subgrid,
				       hipDoubleComplex *fresnel, int subgrid_size, int w_plane){

  

}
				       

/******************************
	  Host Functions
*******************************/

// Doesn't seem like it should be much effort for NVIDIA to add this to CUDA?
// Caveat Emptor: This is a lot slower than C's realloc.
__host__  void *cudaReallocManaged(void *ptr, int size, int size_original){
  
  void *new_ptr;

  //Malloc if passed NULL pointer.
  if(ptr == NULL){
    cudaError_check(hipMallocManaged((void **)&new_ptr, size));
    cudaError_check(hipFree(ptr));
    return new_ptr;
  }

  //Expand our pointers address space. Copy data over.
  if(size > size_original){
    
    cudaError_check(hipMallocManaged((void **)&new_ptr, size));
    cudaError_check(hipMemcpy((void **)&new_ptr, (void **)&ptr,size_original,hipMemcpyDefault));
    cudaError_check(hipFree(ptr));
    return new_ptr;
  }
  //Otherwise shrink our memory space. Bin all data in process.
  else {
    return ptr;
  }
}


__host__ inline double lambda_min(struct bl_data *bl_data, double u) {
    return u * (u < 0 ? bl_data->f_max : bl_data->f_min) / c;
}

__host__ inline double lambda_max(struct bl_data *bl_data, double u) {
    return u * (u < 0 ? bl_data->f_min : bl_data->f_max) / c;
}
 
__host__ inline void fft_shift(hipDoubleComplex *uvgrid, int grid_size) {

  // Shift the FFT
  assert(grid_size % 2 == 0);
  int x, y;
  for (y = 0; y < grid_size; y++) {
    for (x = 0; x < grid_size/2; x++) {
      int ix0 = y * grid_size + x;
      int ix1 = (ix0 + (grid_size+1) * (grid_size/2)) % (grid_size*grid_size);
      hipDoubleComplex temp = uvgrid[ix0];
      uvgrid[ix0] = uvgrid[ix1];
      uvgrid[ix1] = temp;
    }
  }

}

__host__ inline void bin_visibilities(struct vis_data *vis, struct bl_data ***bins,
				      int chunk_count, int wincrement, double theta,
				      int grid_size, int chunk_size){

  std::cout << "Binning our visibilities in U/V for our chunks..\n";
  // Determine bounds in w
  double vis_w_min = 0, vis_w_max = 0;
  int bl;
  for (bl = 0; bl < vis->bl_count; bl++) {
    double w_min = lambda_min(&vis->bl[bl], vis->bl[bl].w_min);
    double w_max = lambda_max(&vis->bl[bl], vis->bl[bl].w_max);
    if (w_min < vis_w_min) { vis_w_min = w_min; }
    if (w_max > vis_w_max) { vis_w_max = w_max; }
  }

  int wp_min = (int) floor(vis_w_min / wincrement + 0.5);
  int wp_max = (int) floor(vis_w_max / wincrement + 0.5);

  // Bin in uv
  int bins_size = sizeof(void *) * chunk_count * chunk_count;
  cudaError_check(hipMallocManaged(&bins, bins_size));
  cudaError_check(hipMemset(bins, 0, bins_size));
    
  int bins_count_size = sizeof(int) * chunk_count * chunk_count;
  int *bins_count;
  cudaError_check(hipMallocManaged(&bins_count, bins_count_size));
  cudaError_check(hipMemset(bins_count, 0, bins_count_size));
  for (bl = 0; bl < vis->bl_count; bl++) {

    // Determine bounds (could be more precise, future work...)
    struct bl_data *bl_data = &vis->bl[bl];
    double u_min = lambda_min(bl_data, bl_data->u_min);
    double u_max = lambda_max(bl_data, bl_data->u_max);
    double v_min = lambda_min(bl_data, bl_data->v_min);
    double v_max = lambda_max(bl_data, bl_data->v_max);

    // Determine first/last overlapping grid chunks
    int cx0 = (floor(u_min * theta + 0.5) + grid_size/2) / chunk_size;
    int cx1 = (floor(u_max * theta + 0.5) + grid_size/2) / chunk_size;
    int cy0 = (floor(v_min * theta + 0.5) + grid_size/2) / chunk_size;
    int cy1 = (floor(v_max * theta + 0.5) + grid_size/2) / chunk_size;

    int cy, cx;
    for (cy = cy0; cy <= cy1; cy++) {
      for (cx = cx0; cx <= cx1; cx++) {
	// Lazy dynamically sized vector

	int bcount = ++bins_count[cy*chunk_count + cx];
	int bcount_p = bcount - bins_count[cy*chunk_count + cx];

	// This is a horrible way of doing this.
	// Why can't NVIDIA re-implement realloc? Also C/C++ is the
	// work of satan (all hail boost::any).
	struct bl_data **bl_data_old = bins[cy*chunk_count + cx];
	cudaError_check(hipMallocManaged(&bins[cy*chunk_count + cx],sizeof(void *) * bcount, hipMemAttachGlobal));
	cudaError_check(hipMemcpy((void **)bins[cy*chunk_count + cx], (void **)bl_data_old,sizeof(void *) * --bcount, hipMemcpyDefault));
	cudaError_check(hipFree((void **)bl_data_old));
	
	bins[cy*chunk_count + cx][bcount-1] = bl_data;

      }
    }
  }
  std::cout << "Bins processed: " << bins_size << "\n";
}


//W-Towers Wrapper.
__host__ hipError_t wtowers_CUDA(const char* visfile, const char* wkernfile, int grid_size,
			   double theta,  double lambda, double bl_min, double bl_max,
				  int subgrid_size, int subgrid_margin, double wincrement){
  //For Benchmarking.
  
  hipError_t error;
  hipEvent_t start, stop;
  float elapsedTime;

  // Load visibility and w-kernel data from HDF5 files.
  
  struct vis_data *vis_dat;
  struct w_kernel_data *wkern_dat;

  cudaError_check(hipMallocManaged((void **)&vis_dat, sizeof(struct vis_data), hipMemAttachGlobal));
  cudaError_check(hipMallocManaged((void **)&wkern_dat, sizeof(struct w_kernel_data), hipMemAttachGlobal));

  int error_hdf5;
  error_hdf5 = load_vis_CUDA(visfile,vis_dat,bl_min,bl_max);
  if (error_hdf5) {
    std::cout << "Failed to Load Visibilities \n";
    return error;
  }
  error_hdf5 = load_wkern_CUDA(wkernfile, theta, wkern_dat);
  if (error_hdf5) {
    std::cout << "Failed to Load W-Kernels \n";
    return error;
  }


  //Allocate our main grid.
  
  int total_gs = grid_size * grid_size;
  
  hipDoubleComplex *grid_dev, *grid_host;
  cudaError_check(hipMalloc((void **)&grid_dev, total_gs * sizeof(hipDoubleComplex)));
  cudaError_check(hipHostMalloc((void **)&grid_host, total_gs * sizeof(hipDoubleComplex)));

  int subgrid_mem_size = sizeof(hipDoubleComplex) * subgrid_size * subgrid_size;


  //Create the fresnel interference pattern for the W-Dimension
  //See Tim Cornwells paper on W-Projection for more information.
  
  hipDoubleComplex *wtransfer;
  cudaError_check(hipMallocManaged((void **)&wtransfer, subgrid_mem_size, hipMemAttachGlobal));

  int x,y;
  for (y=0; y < subgrid_size; ++y){

    for (x=0; x < subgrid_size; ++x){

      double l = theta * (double)(x - subgrid_size / 2) / subgrid_size;
      double m = theta * (double)(y - subgrid_size / 2) / subgrid_size;
      double ph = wincrement * (1 - sqrt(1 - l*l - m*m));

      hipDoubleComplex wtrans = make_hipDoubleComplex(0, 2 * M_PI * ph);
      wtransfer[y * subgrid_size + x] = cu_cexp_d(wtrans);
    }

  }

  //Create FFT Plans for our frequent fft's.

  hipfftHandle fft_plan;
  hipfftPlan2d(&fft_plan,subgrid_size,subgrid_size,HIPFFT_D2Z);


  //Allocate subgrids/subimgs on the GPU
  
  assert( grid_size % subgrid_size == 0);
  int chunk_count_1d = grid_size / subgrid_size;
  int total_chunks = chunk_count_1d * chunk_count_1d;

  hipDoubleComplex **subgrids, **subimgs;

  cudaError_check(hipMallocManaged(&subgrids, total_chunks * sizeof(hipDoubleComplex)));
  cudaError_check(hipMallocManaged(&subimgs, total_chunks * sizeof(hipDoubleComplex)));

  //Create streams for each tower and allocate our chunks on GPU memory.
  
  hipStream_t streams[total_chunks];
  for(int i = 0; i < total_chunks; ++i){

    hipStreamCreate(&streams[i]);

    cudaError_check(hipMallocManaged(subgrids + i, subgrid_mem_size * sizeof(hipDoubleComplex)));
    cudaError_check(hipMallocManaged(subimgs + i, subgrid_mem_size * sizeof(hipDoubleComplex)));

  }

  struct bl_data ***bins;

  bin_visibilities(vis_dat, bins, chunk_count_1d, wincrement, theta, grid_size, subgrid_size);

  
  return error;

}
