//C++ Includes
#include <iostream>
#include <fstream>
#include <cstdlib>

//CUDA Includes
#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>
#include "hip/hip_runtime.h"
#include "math.h"
#include "hip/hip_runtime_api.h"

//Our Include
#include "wtowers_common.h"



//W-Towers Wrapper.
__host__ hipError_t wtowers_host(const char* visfile, const char* wkernfile, int grid_size,
			   double theta,  double lambda, double bl_min, double bl_max,
				  int iter){

  hipError_t error;
  
  return error;

}
