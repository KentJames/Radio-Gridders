#include "hip/hip_runtime.h"
//C++ Includes
#include <iostream>
#include <fstream>
#include <cstdlib>
#include <cassert>

//CUDA Includes
#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>
#include "hip/hip_runtime.h"
#include "math.h"
#include "hip/hip_runtime_api.h"

//Our Include
#include "wtowers_common.h"


/*****************************
      CUDA Error Checker
******************************/
 
#define cudaError_check(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){

  if (code != hipSuccess){
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

#define cuFFTError_check(ans) { cufftAssert((ans), __FILE__, __LINE__); }
inline void cufftAssert(hipfftResult code, const char *file, int line, bool abort=true){

  if (code != HIPFFT_SUCCESS){
    fprintf(stderr,"cufftAssert: %d %s %d\n", code, file, line);
    if (abort) exit(code);
  }
}

/*****************************
        Device Functions
 *****************************/

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600


#else //Pre-pascal devices.

__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

#endif

// Complex functions that I wished were implemented...
// Sometimes I feel NVIDIA's Complex library is a bit half finished.

__host__ __device__ inline hipDoubleComplex cu_cexp_d (hipDoubleComplex z){

  hipDoubleComplex res;
  double t = exp (z.x);
  sincos (z.y, &res.y, &res.x);
  res.x *= t;
  res.y *= t;
  return res;

}

//Stolen from Peter Wortmann (who stole it from Stack Overflow)
__host__ __device__ inline hipDoubleComplex cu_cpow(hipDoubleComplex base, int exp){

  hipDoubleComplex result = make_hipDoubleComplex(1.0,1.0);
  //Can't recurse on a device function!!!
  //  if (exp < 0) return hipCdiv(make_hipDoubleComplex(1.0,1.0), cu_cpow(base, -exp));
  if (exp < 0) return base; 
  if (exp == 1) return base;
  while (exp){
    if (exp & 1) result = hipCmul(base,result);
    exp >>= 1;
    base = hipCmul(base,base);
  }
  return result;
}


//Gets minimum/maximum co-ordinate in a particular baseline.
__host__ __device__ inline double lambda_min(struct bl_data *bl_data, double u) {
    return u * (u < 0 ? bl_data->f_max : bl_data->f_min) / c;
}

__host__ __device__ inline double lambda_max(struct bl_data *bl_data, double u) {
    return u * (u < 0 ? bl_data->f_min : bl_data->f_max) / c;
}


__host__ __device__ inline static double uvw_lambda(struct bl_data *bl_data,
				  int time, int freq, int uvw) {
    return bl_data->uvw[3*time+uvw] * bl_data->freq[freq] / c;
    
  }

__host__ __device__ inline static void frac_coord_flat(int grid_size, int kernel_size, int oversample,
                              double theta,
                              struct flat_vis_data *vis,
                              int i,
                              double d_u, double d_v,
                              int *grid_offset, int *sub_offset) {
#ifdef ASSUME_UVW_0
    double x = 0, y = 0;
#else
    double x = theta * (vis->u[i] - d_u);
    double y = theta * (vis->v[i] - d_v);
#endif
    int flx = (int)floor(x + .5 / oversample);
    int fly = (int)floor(y + .5 / oversample);
    int xf = (int)floor((x - (double)flx) * oversample + .5);
    int yf = (int)floor((y - (double)fly) * oversample + .5);
    *grid_offset =
        (fly+grid_size/2-kernel_size/2)*grid_size +
        (flx+grid_size/2-kernel_size/2);
    *sub_offset = kernel_size * kernel_size * (yf*oversample + xf);
}


__host__ __device__ inline static void frac_coord(int grid_size, int kernel_size, int oversample,
                              double theta,
                              struct bl_data *bl_data,
                              int time, int freq,
                              double d_u, double d_v,
                              int *grid_offset, int *sub_offset) {
#ifdef ASSUME_UVW_0
    double x = 0, y = 0;
#else
    double x = theta * (uvw_lambda(bl_data, time, freq, 0) - d_u);
    double y = theta * (uvw_lambda(bl_data, time, freq, 1) - d_v);
#endif
    int flx = (int)floor(x + .5 / oversample);
    int fly = (int)floor(y + .5 / oversample);
    int xf = (int)floor((x - (double)flx) * oversample + .5);
    int yf = (int)floor((y - (double)fly) * oversample + .5);
    *grid_offset =
        (fly+grid_size/2-kernel_size/2)*grid_size +
        (flx+grid_size/2-kernel_size/2);
    *sub_offset = kernel_size * kernel_size * (yf*oversample + xf);
}


//From Kyrills implementation in SKA/RC
__device__ inline void scatter_grid_add(hipDoubleComplex *uvgrid, int grid_size, int grid_pitch,
					int grid_point_u, int grid_point_v, hipDoubleComplex sum){

  
  // Atomically add to grid. This is the bottleneck of this kernel.
  if (grid_point_u < 0 || grid_point_u >= grid_size ||
      grid_point_v < 0 || grid_point_v >= grid_size)
    return;

  // Bottom half? Mirror
  //if (grid_point_u >= grid_size / 2) {
  //  grid_point_v = grid_size - grid_point_v - 1;
  //  grid_point_u = grid_size - grid_point_u - 1;
  //}

  // Add to grid. This is the bottleneck of the entire kernel
  atomicAdd(&uvgrid[grid_point_u + grid_pitch*grid_point_v].x, sum.x);
  atomicAdd(&uvgrid[grid_point_u + grid_pitch*grid_point_v].y, sum.y);

}

//Scatters grid points from a non-hierarchical dataset.
//Advantage: Locality is almost certainly better for fragmented datasets.
//Disadvantage: Not able to do baseline specific calibration, such as ionosphere correction.
__device__ inline void scatter_grid_point_flat(
					  struct flat_vis_data *vis, // Our bins of UV Data
					  hipDoubleComplex *uvgrid, // Our main UV Grid
					  struct w_kernel_data *wkern, //Our W-Kernel
					  int max_supp, // Max size of W-Kernel
					  int myU, //Our assigned u/v points.
					  int myV, // ^^^
					  double wstep, // W-Increment 
					  int subgrid_size, //The size of our w-towers subgrid.
					  double theta, // Field of View Size
					  int offset_u, // Offset from top left of main grid to t.l of subgrid.
					  int offset_v, // ^^^^
					  int offset_w,
					  double3 u_rng,
					  double3 v_rng,
					  double3 w_rng
					  ){ 

  int grid_point_u = myU, grid_point_v = myV;
  hipDoubleComplex sum  = make_hipDoubleComplex(0.0,0.0);

  short supp = short(wkern->size_x);
  
  //  for (int i = 0; i < visibilities; i++) {
  int vi;
  for (vi = 0; vi < vis->number_of_vis; ++vi){
    
    if(vis->u[vi] < u_rng.x ||
       vis->u[vi] >= u_rng.y ||
       vis->v[vi] < v_rng.x ||
       vis->v[vi]  >= v_rng.y ||
       vis->w[vi] < w_rng.x ||
       vis->w[vi]  >= w_rng.y) {
      continue;//Skip
    }
    
    
    //double u = vis->u[vi];
    //double v = vis->v[vi];
    double w = vis->w[vi];
    int w_plane = fabs((w - wkern->w_min) / (wkern->w_step + .5));
    int grid_offset, sub_offset;
    frac_coord_flat(subgrid_size, wkern->size_x, wkern->oversampling,
		    theta, vis, vi, offset_u, offset_v, &grid_offset, &sub_offset);
    int u = grid_offset % subgrid_size; 
    int v = grid_offset / subgrid_size;

    // Determine convolution point. This is basically just an
    // optimised way to calculate
    //   myConvU = (myU - u) % max_supp
    //   myConvV = (myV - v) % max_supp
    //	int2 xy = getcoords_xy(u,v,subgrid_size,theta,max_supp);
    int myConvU = (u - myU) % max_supp;
    int myConvV = (v - myV) % max_supp;
    if (myConvU < 0) myConvU += max_supp;
    if (myConvV < 0) myConvV += max_supp;

    // Determine grid point. Because of the above we know here that
    //   myGridU % max_supp = myU
    //   myGridV % max_supp = myV
    int myGridU = u + myConvU
      , myGridV = v + myConvV;

    // Grid point changed?
    if (myGridU != grid_point_u || myGridV != grid_point_v) {
      // Atomically add to grid. This is the bottleneck of this kernel.
      scatter_grid_add(uvgrid, subgrid_size, subgrid_size, grid_point_u, grid_point_v, sum);
      // Switch to new point
      sum = make_hipDoubleComplex(0.0, 0.0);
      grid_point_u = myGridU;
      grid_point_v = myGridV;
    }
    //TODO: Re-do the w-kernel/gcf for our data.
    //	hipDoubleComplex px;
    hipDoubleComplex px = *(hipDoubleComplex*)&wkern->kern_by_w[w_plane].data[sub_offset + myConvU * supp + myConvV];	
    // Sum up
    hipDoubleComplex vi_v = *(hipDoubleComplex*)&vis->vis[vi];
    sum = hipCfma(px, vi_v, sum);
      
    
  }

  // Add remaining sum to grid
  scatter_grid_add(uvgrid, subgrid_size, subgrid_size, grid_point_u, grid_point_v, sum);

}



//From Kyrills Implementation in SKA/RC. Modified to suit our data format.
//Assumes pre-binned (in u/v) data
__device__ inline void scatter_grid_point(
					  struct vis_data *bin, // Our bins of UV Data
					  hipDoubleComplex *uvgrid, // Our main UV Grid
					  struct w_kernel_data *wkern, //Our W-Kernel
					  int max_supp, // Max size of W-Kernel
					  int myU, //Our assigned u/v points.
					  int myV, // ^^^
					  double wstep, // W-Increment 
					  int subgrid_size, //The size of our w-towers subgrid.
					  double theta, // Field of View Size
					  int offset_u, // Offset from top left of main grid to t.l of subgrid.
					  int offset_v, // ^^^^
					  int offset_w,
					  double3 u_rng,
					  double3 v_rng,
					  double3 w_rng
					  ){ 

  int grid_point_u = myU, grid_point_v = myV;
  hipDoubleComplex sum  = make_hipDoubleComplex(0.0,0.0);

  short supp = short(wkern->size_x);
  
  //  for (int i = 0; i < visibilities; i++) {
  int bl, time, freq;
  for (bl = 0; bl < bin->bl_count; ++bl){
    struct bl_data *bl_d = &bin->bl[bl];

    //Keep this for now. It reduces performance by 50%.
    // TODO: Bounds check elsewhere.
    if(lambda_max(bl_d, bl_d->u_max) < u_rng.x ||
       lambda_min(bl_d, bl_d->u_min) >= u_rng.y ||
       lambda_max(bl_d, bl_d->v_max) < v_rng.x ||
       lambda_min(bl_d, bl_d->v_min) >= v_rng.y ||
       lambda_max(bl_d, bl_d->w_max) < w_rng.x ||
       lambda_min(bl_d, bl_d->w_min) >= w_rng.y) {
      continue;//Skip
    }
    
    
    for (time = 0; time < bl_d->time_count; ++time){
      for(freq = 0; freq < bl_d->freq_count; ++freq){
	// Load pre-calculated positions
	//int u = uvo[i].u, v = uvo[i].v;
	//	int u = (int)uvw_lambda(bl_d, time, freq, 0);
	//int v = (int)uvw_lambda(bl_d, time, freq, 1);
	double w = uvw_lambda(bl_d, time, freq, 2) - offset_w;
	int w_plane = fabs((w - wkern->w_min) / (wkern->w_step + .5));
	int grid_offset, sub_offset;
	frac_coord(subgrid_size, wkern->size_x, wkern->oversampling,
		   theta, bl_d, time, freq, offset_u, offset_v, &grid_offset, &sub_offset);
	int u = grid_offset % subgrid_size; 
	int v = grid_offset / subgrid_size;

	// Determine convolution point. This is basically just an
	// optimised way to calculate
	//   myConvU = (myU - u) % max_supp
	//   myConvV = (myV - v) % max_supp
	//	int2 xy = getcoords_xy(u,v,subgrid_size,theta,max_supp);
	int myConvU = (u - myU) % max_supp;
	int myConvV = (v - myV) % max_supp;
	if (myConvU < 0) myConvU += max_supp;
	if (myConvV < 0) myConvV += max_supp;

	// Determine grid point. Because of the above we know here that
	//   myGridU % max_supp = myU
	//   myGridV % max_supp = myV
	int myGridU = u + myConvU
	  , myGridV = v + myConvV;

	// Grid point changed?
	if (myGridU != grid_point_u || myGridV != grid_point_v) {
	  // Atomically add to grid. This is the bottleneck of this kernel.
	  scatter_grid_add(uvgrid, subgrid_size, subgrid_size, grid_point_u, grid_point_v, sum);
	  // Switch to new point
	  sum = make_hipDoubleComplex(0.0, 0.0);
	  grid_point_u = myGridU;
	  grid_point_v = myGridV;
	  }
	//TODO: Re-do the w-kernel/gcf for our data.
	//	hipDoubleComplex px;
	hipDoubleComplex px = *(hipDoubleComplex*)&wkern->kern_by_w[w_plane].data[sub_offset + myConvU * supp + myConvV];	
	// Sum up
	hipDoubleComplex vi = *(hipDoubleComplex*)&bl_d->vis[time*bl_d->freq_count+freq];
	sum = hipCfma(px, vi, sum);
      }
    }
  }

  // Add remaining sum to grid
  scatter_grid_add(uvgrid, subgrid_size, subgrid_size, grid_point_u, grid_point_v, sum);

}


/******************************
            Kernels
*******************************/


//Elementwise multiplication of subimg with fresnel. 
__global__ void fresnel_subimg_mul(hipDoubleComplex *subgrid,
				   hipDoubleComplex *fresnel,
				   hipDoubleComplex *subimg,
				   int n,
				   int wp){

  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if(x < n && y < n){

    hipDoubleComplex wtrans = cu_cpow(fresnel[y * n + x], wp);
    subimg[y * n + x] = hipCmul(fresnel[y * n + x], subimg[y * n +x]);
    subimg[y * n + x] = hipCadd(subimg[y * n + x], subgrid[y * n + x]);
    subgrid[y * n + x] = make_hipDoubleComplex(0.0,0.0);
  }

}

//Set the total grid size to cover every pixel in the main grid.
__global__ void add_subs2main_kernel(hipDoubleComplex *main, hipDoubleComplex *subs,
				     int main_size, int sub_size, int sub_margin,
				     int chunk_count, int chunk_size){


  int x = (blockDim.x * blockIdx.x + threadIdx.x) - main_size/2;
  int y = (blockDim.y * blockIdx.y + threadIdx.y) - main_size/2;
  //  int ts = chunk_count * chunk_count  * sub_size * sub_size;
  for(int cy = 0; cy < chunk_count; ++cy){
    for(int cx = 0; cx < chunk_count; ++cx){
      
      int x_min = chunk_size*cx - main_size/2; //- sub_size/2;
      int y_min = chunk_size*cy - main_size/2; //- sub_size/2;
      
      //int x_max = sub_*(cx+1);
      //int y_max = sub_size*(cy+1);

      int x0 = x_min - sub_margin/2;
      int y0 = y_min - sub_margin/2;

      int x1 = x0 + sub_size;
      int y1 = y0 + sub_size;

      if (x0 < -main_size/2) { x0 = -main_size/2; }
      if (y0 < -main_size/2) { y0 = -main_size/2; }
      if (x1 > main_size/2) { x1 = main_size/2; }
      if (y1 > main_size/2) { y1 = main_size/2; }
      hipDoubleComplex *main_mid = main + (main_size+1)*main_size/2;
      if(y>= y0 && y < y1 && x>= x0 && x < x1){
	
	int y_s = y - y_min + sub_margin / 2;
	int x_s = x - x_min + sub_margin / 2;
	main_mid[y*main_size + x] = hipCadd(main_mid[y*main_size+x],
				       (subs+(((cy*chunk_count)+cx)*sub_size*sub_size))
					   [y_s*sub_size + x_s]);
	main_mid[y*main_size + x] = hipCdiv(main_mid[y*main_size + x],
					   make_hipDoubleComplex(sub_size * sub_size,
								0.0));
	//Not sure if this is good style. 1) Calculate offset. 2) Dereference via array notation
      }
    }
  }
  
}

//Transforms grid to w==0 plane.
__global__ void w0_transfer_kernel(hipDoubleComplex *grid, hipDoubleComplex *base, int exp, int size){

  int x = blockIdx.x * blockDim.x + threadIdx.x;

  if(x<size*size) grid[x] = hipCdiv(grid[x],cu_cpow(base[x],exp));
}

//Shifts a 2D grid to be in the right place for an FFT. 
__global__ void fft_shift_kernel(hipDoubleComplex *grid, int size){

  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if(x<size/2 && y <size){

    int ix0 = y * size + x;
    int ix1 = (ix0 + (size + 1) * (size/2)) % (size*size);

    hipDoubleComplex temp = grid[ix0];
    grid[ix0] = grid[ix1];
    grid[ix1] = temp;

  }

}

//This is our Romein-style scatter gridder. Works on flat visibility data.
__global__ void scatter_grid_kernel_flat(
					 struct flat_vis_data *vis, // No. of visibilities
					 struct w_kernel_data *wkern, // No. of wkernels
					 hipDoubleComplex *uvgrid, //Our UV-Grid
					 int max_support, //  Convolution size
					 int subgrid_size, // Subgrid size
					 double wstep, // W-Increment
					 double theta, // Field of View
					 int offset_u, // Top left offset from top left main grid
					 int offset_v, // ^^^^
					 int offset_w,
					 double3 u_rng,
					 double3 v_rng,
					 double3 w_rng
					 ){

  //Assign some visibilities to grid;

  

  
  for(int i = threadIdx.x; i < max_support * max_support; i += blockDim.x){
    //  int i = threadIdx.x + blockIdx.x * blockDim.x;
    int myU = i % max_support;
    int myV = i / max_support;

    scatter_grid_point_flat(vis+blockIdx.x, uvgrid, wkern, max_support, myU, myV, wstep,
			    subgrid_size, theta, offset_u, offset_v, offset_w,
			    u_rng, v_rng, w_rng);
		       
  }
}


//This is our Romein-style scatter gridder. Works on hierarchical visibility data (bl->time->freq).
__global__ void scatter_grid_kernel(struct vis_data *bin, // Baseline bin
				    struct w_kernel_data *wkern, // No. of wkernels
				    hipDoubleComplex *uvgrid, //Our UV-Grid
				    int max_support, //  Convolution size
				    int subgrid_size, // Subgrid size
				    double wstep, // W-Increment
				    double theta, // Field of View
				    int offset_u, // Top left offset from top left main grid
				    int offset_v, // ^^^^
				    int offset_w,
				    double3 u_rng,
				    double3 v_rng,
				    double3 w_rng				   
				    ){
  
  for(int i = threadIdx.x; i < max_support * max_support; i += blockDim.x){
    //  int i = threadIdx.x + blockIdx.x * blockDim.x;
    int myU = i % max_support;
    int myV = i / max_support;

    scatter_grid_point(bin, uvgrid, wkern, max_support, myU, myV, wstep,
		       subgrid_size, theta, offset_u, offset_v, offset_w,
		       u_rng, v_rng, w_rng);
		       
  }
}

/******************************
	  Host Functions
*******************************/

__host__ inline void init_grid_zero(hipDoubleComplex *uvgrid, int grid_size){

  for(int x = 0; x< grid_size; ++x){
    for(int y = 0; y< grid_size; ++y){
      *(uvgrid+x*grid_size+y) = make_hipDoubleComplex(0.0, 0.0);
    }
  }

}

// Get coarse-grained co-ordinate.
__host__ inline static int coord(int grid_size, double theta,
                 struct bl_data *bl_data,
                 int time, int freq) {
#ifdef ASSUME_UVW_0
    int x = 0, y = 0;
#else
    int x = (int)floor(theta * uvw_lambda(bl_data, time, freq, 0) + .5);
    int y = (int)floor(theta * uvw_lambda(bl_data, time, freq, 1) + .5);
#endif
    return (y+grid_size/2) * grid_size + (x+grid_size/2);
}

// Get coarse-grained co-ordinate.
__host__ inline static int coord_flat(int grid_size, double theta,
                 struct flat_vis_data *vis_data,
                 int vi) {
#ifdef ASSUME_UVW_0
    int x = 0, y = 0;
#else
    int x = (int)floor(theta * vis_data->u[vi] + .5);
    int y = (int)floor(theta * vis_data->v[vi] + .5);
#endif
    return (y+grid_size/2) * grid_size + (x+grid_size/2);
}

// Uniformly weights all visibilities.
__host__ inline void weight(unsigned int *wgrid, int grid_size, double theta,
            struct vis_data *vis) {

  int total_vis=0;
    // Simple uniform weighting
  int bl, time, freq;
    memset(wgrid, 0, grid_size * grid_size * sizeof(unsigned int));
    for (bl = 0; bl < vis->bl_count; bl++) {
        for (time = 0; time < vis->bl[bl].time_count; time++) {
            for (freq = 0; freq < vis->bl[bl].freq_count; freq++) {
                wgrid[coord(grid_size, theta, &vis->bl[bl], time, freq)]++;
		++total_vis;
            }
        }
    }

    
    for (bl = 0; bl < vis->bl_count; bl++) {
        for (time = 0; time < vis->bl[bl].time_count; time++) {
            for (freq = 0; freq < vis->bl[bl].freq_count; freq++) {
                vis->bl[bl].vis[time*vis->bl[bl].freq_count + freq]
                    /= wgrid[coord(grid_size, theta, &vis->bl[bl], time, freq)];
            }
        }
    }

}

// Uniformly weights all visibilities on a flat structre.
__host__ inline void weight_flat(unsigned int *wgrid, int grid_size, double theta,
            struct flat_vis_data *vis) {

    // Simple uniform weighting

    memset(wgrid, 0, grid_size * grid_size * sizeof(unsigned int));
    int vii;

    for (vii = 0; vii<vis->number_of_vis; ++vii){
      wgrid[coord_flat(grid_size, theta, vis, vii)]++;
    }

    for (vii = 0; vii<vis->number_of_vis; ++vii){
      vis->vis[vii] /= wgrid[coord_flat(grid_size, theta, vis, vii)];
    }
      
}


//Shifts middle of image to top left corner, to make sure FFT is correct.
// (Remember to use this again after the FFT too...)
__host__ inline void fft_shift(hipDoubleComplex *uvgrid, int grid_size) {

  // Shift the FFT
  assert(grid_size % 2 == 0);
  int x, y;
  for (y = 0; y < grid_size; y++) {
    for (x = 0; x < grid_size/2; x++) {
      int ix0 = y * grid_size + x;
      int ix1 = (ix0 + (grid_size+1) * (grid_size/2)) % (grid_size*grid_size);
      hipDoubleComplex temp = uvgrid[ix0];
      uvgrid[ix0] = uvgrid[ix1];
      uvgrid[ix1] = temp;
    }
  }
}

//Ensures 2-D array is hermitian symmetric.
__host__ inline void make_hermitian(hipDoubleComplex *uvgrid, int grid_size){

  hipDoubleComplex *p0;

  if (grid_size % 2 == 0) {
    p0 = uvgrid + grid_size + 1;
  }
  else {
    p0 = uvgrid;
  }

  hipDoubleComplex *p1 = uvgrid + grid_size * grid_size - 1;

  while (p0 < p1) {
    hipDoubleComplex g0 = *p0;

    hipCadd(*p0++,hipConj(*p1));
    hipCadd(*p1--,hipConj(g0));
    //    *p0++ += hipConj(*p1);
    //*p1-- += hipConj(g0);
  }

  assert ( p0 == p1 && p0 == uvgrid + (grid_size + 1) * (grid_size/2));
  hipCadd(*p0,hipConj(*p0));
  //  *p0 += hipConj(*p0);


}

//Splits our visibilities up into contiguous bins, for each block to apply.
__host__ inline void bin_flat_visibilities(struct flat_vis_data *vis_bins,
					   struct flat_vis_data *vis,
					   int blocks){

  std::cout << "Binning Visibilities. No. of vis: " << vis->number_of_vis << " No. of Blocks: " << blocks << "\n";
  
  int vis_per_block = vis->number_of_vis / blocks;
  int leftovers = vis->number_of_vis % blocks;


  int i;
  for(i = 0; i < blocks-1; ++i){

    cudaError_check(hipMallocManaged((void**)&(vis_bins+i)->u,
				      sizeof(double) * vis_per_block, hipMemAttachGlobal));
    cudaError_check(hipMallocManaged((void**)&(vis_bins+i)->v,
				      sizeof(double) * vis_per_block, hipMemAttachGlobal));
    cudaError_check(hipMallocManaged((void**)&(vis_bins+i)->w,
				      sizeof(double) * vis_per_block, hipMemAttachGlobal));
    cudaError_check(hipMallocManaged((void**)&(vis_bins+i)->vis,
				      sizeof(double _Complex) * vis_per_block, hipMemAttachGlobal));


    cudaError_check(hipMemcpy((vis_bins+i)->u, vis->u + vis_per_block * i,
			       sizeof(double) * vis_per_block, hipMemcpyDefault));
    cudaError_check(hipMemcpy((vis_bins+i)->v, vis->v + vis_per_block * i,
			       sizeof(double) * vis_per_block, hipMemcpyDefault));
    cudaError_check(hipMemcpy((vis_bins+i)->w, vis->w + vis_per_block * i,
			       sizeof(double) * vis_per_block, hipMemcpyDefault));
    cudaError_check(hipMemcpy((vis_bins+i)->vis, vis->vis + vis_per_block * i,
			       sizeof(double _Complex) * vis_per_block, hipMemcpyDefault));
    (vis_bins+i)->number_of_vis = vis_per_block;
  }
  
  //Last one gets remainders.


  cudaError_check(hipMallocManaged((void**)&(vis_bins+i)->u,
				    sizeof(double) * (vis_per_block + leftovers), hipMemAttachGlobal));
  cudaError_check(hipMallocManaged((void**)&(vis_bins+i)->v,
				    sizeof(double) * (vis_per_block + leftovers), hipMemAttachGlobal));
  cudaError_check(hipMallocManaged((void**)&(vis_bins+i)->w,
				    sizeof(double) * (vis_per_block + leftovers), hipMemAttachGlobal));
  cudaError_check(hipMallocManaged((void**)&(vis_bins+i)->vis,
				    sizeof(double _Complex) * (vis_per_block + leftovers), hipMemAttachGlobal));
    

  
  cudaError_check(hipMemcpy((vis_bins+i)->u, vis->u + vis_per_block * i,
			     sizeof(double) * (vis_per_block+leftovers), hipMemcpyDefault));
  cudaError_check(hipMemcpy((vis_bins+i)->v, vis->v + vis_per_block * i,
			     sizeof(double) * (vis_per_block+leftovers), hipMemcpyDefault));
  cudaError_check(hipMemcpy((vis_bins+i)->w, vis->w + vis_per_block * i,
			     sizeof(double) * (vis_per_block+leftovers), hipMemcpyDefault));
  cudaError_check(hipMemcpy((vis_bins+i)->vis, vis->vis + vis_per_block * i,
			     sizeof(double _Complex) * (vis_per_block+leftovers), hipMemcpyDefault));
   (vis_bins+i)->number_of_vis = vis_per_block + leftovers;

  
}


//Bins visibilities in u/v for w-towers style subgrids.
__host__ inline void bin_visibilities(struct vis_data *vis, struct vis_data *bins,
				      int chunk_count, int wincrement, double theta,
				      int grid_size, int chunk_size, int *w_min, int *w_max){

  std::cout << "Binning our visibilities in U/V for our chunks..\n";
  // Determine bounds in w
  double vis_w_min = 0, vis_w_max = 0;
  int bl;
  for (bl = 0; bl < vis->bl_count; bl++) {
    double w_min = lambda_min(&vis->bl[bl], vis->bl[bl].w_min);
    double w_max = lambda_max(&vis->bl[bl], vis->bl[bl].w_max);
    if (w_min < vis_w_min) { vis_w_min = w_min; }
    if (w_max > vis_w_max) { vis_w_max = w_max; }
  }

  int wp_min = (int) floor(vis_w_min / wincrement + 0.5);
  int wp_max = (int) floor(vis_w_max / wincrement + 0.5);

  *w_min = wp_min; // Report w-values back to calling function
  *w_max = wp_max;

  // Bin in uv
  int bins_size = sizeof(struct vis_data) * chunk_count * chunk_count;
  //cudaError_check(hipMallocManaged(&bins, bins_size, hipMemAttachGlobal));
  //cudaError_check(hipMemset(bins, 0, bins_size));
  
  int bins_count_size = sizeof(int) * chunk_count * chunk_count;
  int *bins_count;
  cudaError_check(hipMallocManaged(&bins_count, bins_count_size, hipMemAttachGlobal));
  cudaError_check(hipMemset(bins_count, 0, bins_count_size));
  for (bl = 0; bl < vis->bl_count; bl++) {
    
    // Determine bounds (could be more precise, future work...)
    struct bl_data *bl_data = &vis->bl[bl];
    //cudaError_check(hipMallocManaged(&bl_data, sizeof(struct bl_data), hipMemAttachGlobal));
    //bl_data = &vis->bl[bl];
    double u_min = lambda_min(bl_data, bl_data->u_min);
    double u_max = lambda_max(bl_data, bl_data->u_max);
    double v_min = lambda_min(bl_data, bl_data->v_min);
    double v_max = lambda_max(bl_data, bl_data->v_max);
    
    // Determine first/last overlapping grid chunks
    int cx0 = (floor(u_min * theta + 0.5) + grid_size/2) / chunk_size;
    int cx1 = (floor(u_max * theta + 0.5) + grid_size/2) / chunk_size;
    int cy0 = (floor(v_min * theta + 0.5) + grid_size/2) / chunk_size;
    int cy1 = (floor(v_max * theta + 0.5) + grid_size/2) / chunk_size;
    
    int cy, cx;
    for (cy = cy0; cy <= cy1; cy++) {
      for (cx = cx0; cx <= cx1; cx++) {
	// Lazy dynamically sized vector
	
	int bcount = ++bins[cy*chunk_count + cx].bl_count;
	int bcount_p = bcount - 1;
	
	struct bl_data *bl_data_old = bins[cy*chunk_count+cx].bl;
	struct bl_data *temp;
	
	cudaError_check(hipMallocManaged(&temp,
					  sizeof(struct bl_data) * bcount));
	cudaError_check(hipMemcpy(temp, bl_data_old,
				   sizeof(struct bl_data)*(bcount-1),
				   hipMemcpyDefault));
	cudaError_check(hipMemcpy(temp+bcount_p, bl_data,
				   sizeof(struct bl_data),hipMemcpyDefault));

	bins[cy*chunk_count + cx].bl = temp;
	cudaError_check(hipFree(bl_data_old));
	//	cudaError_check(hipFree(temp));
						
      }
    }
  }
  std::cout << "Bins processed: " << bins_size << "\n";
}

//W-Towers Wrapper.
__host__ hipError_t wtowers_CUDA(const char* visfile, const char* wkernfile, int grid_size,
			   double theta,  double lambda, double bl_min, double bl_max,
				  int subgrid_size, int subgrid_margin, double wincrement){

  //API Variables
  hipError_t error = (hipError_t)0;
  
 
  //For Benchmarking.
  hipEvent_t start, stop;
  float elapsedTime;

  // Load visibility and w-kernel data from HDF5 files.
  struct vis_data *vis_dat;
  struct w_kernel_data *wkern_dat;

  cudaError_check(hipMallocManaged((void **)&vis_dat, sizeof(struct vis_data), hipMemAttachGlobal));
  cudaError_check(hipMallocManaged((void **)&wkern_dat, sizeof(struct w_kernel_data), hipMemAttachGlobal));

  int error_hdf5;
  error_hdf5 = load_vis_CUDA(visfile,vis_dat,bl_min,bl_max);
  if (error_hdf5) {
    std::cout << "Failed to Load Visibilities \n";
    return error;
  }
  error_hdf5 = load_wkern_CUDA(wkernfile, theta, wkern_dat);
  if (error_hdf5) {
    std::cout << "Failed to Load W-Kernels \n";
    return error;
  }

  // Work out our minimum and maximum w-planes.

  double vis_w_min = 0, vis_w_max = 0;
  for (int bl = 0; bl < vis_dat->bl_count; ++bl){
    double w_min = lambda_min(&vis_dat->bl[bl], vis_dat->bl[bl].w_min);
    double w_max = lambda_max(&vis_dat->bl[bl], vis_dat->bl[bl].w_max);
    if (w_min < vis_w_min) { vis_w_min = w_min; }
    if (w_max > vis_w_max) { vis_w_max = w_max; }
  }
  int wp_min = (int) floor(vis_w_min / wincrement + 0.5);
  int wp_max = (int) floor(vis_w_max / wincrement + 0.5);
  std::cout << "Our W-Plane Min/Max: " << wp_min << " " << wp_max << "\n";


  
  //Allocate our main grid.
  
  int total_gs = grid_size * grid_size;
  
  hipDoubleComplex *grid_dev, *grid_host;
  cudaError_check(hipMalloc((void **)&grid_dev, total_gs * sizeof(hipDoubleComplex)));
  cudaError_check(hipHostMalloc((void **)&grid_host, total_gs * sizeof(hipDoubleComplex)));





  //Create the fresnel interference pattern for the W-Dimension
  //Can make this a kernel.
  //See Tim Cornwells paper on W-Projection for more information.
  int subgrid_mem_size = sizeof(hipDoubleComplex) * subgrid_size * subgrid_size;  
  hipDoubleComplex *wtransfer;
  cudaError_check(hipMallocManaged((void **)&wtransfer, subgrid_mem_size, hipMemAttachGlobal));

  int x,y;
  for (y=0; y < subgrid_size; ++y){
    for (x=0; x < subgrid_size; ++x){
      double l = theta * (double)(x - subgrid_size / 2) / subgrid_size;
      double m = theta * (double)(y - subgrid_size / 2) / subgrid_size;
      double ph = wincrement * (1 - sqrt(1 - l*l - m*m));
      hipDoubleComplex wtrans = make_hipDoubleComplex(0.0, 2 * M_PI * ph);
      wtransfer[y * subgrid_size + x] = cu_cexp_d(wtrans);
    }
  }
  fft_shift(wtransfer, subgrid_size);

  //Allocate subgrids/subimgs on the GPU
  
  assert( grid_size % subgrid_size == 0);
  
  int chunk_size = subgrid_size - subgrid_margin;
  int chunk_count_1d = grid_size / chunk_size + 1;
  int total_chunks = chunk_count_1d * chunk_count_1d;

  //Allocate all our subgrids/subimgs contiguously.
  hipDoubleComplex *subgrids, *subimgs;
  
  cudaError_check(hipMallocManaged((void **)&subgrids,
				    total_chunks * subgrid_mem_size  * sizeof(hipDoubleComplex),
				    hipMemAttachGlobal));
  cudaError_check(hipMallocManaged((void **)&subimgs,
				    total_chunks * subgrid_mem_size * sizeof(hipDoubleComplex),
				    hipMemAttachGlobal));

  //  cudaError_check(hipMemset(subimgs, 1.2, total_chunks * subgrid_mem_size * sizeof(hipDoubleComplex)));
  //Create streams for each tower and allocate our chunks in unified memory.
  //Also set our FFT plans while we are here.
  //Initialise cublas handle. We use cublas to multiply our fresnel phase screen.

  hipStream_t *streams = (hipStream_t *) malloc(total_chunks * sizeof(hipStream_t));
  hipfftHandle *subgrid_plans = (hipfftHandle *) malloc(total_chunks * sizeof(hipfftHandle));
  
  for(int i = 0; i < total_chunks; ++i){

    //Create stream.
    cudaError_check(hipStreamCreate(&streams[i]));

    //Assign FFT Plan to each stream
    cuFFTError_check(hipfftPlan2d(&subgrid_plans[i],subgrid_size,subgrid_size,HIPFFT_Z2Z));
    cuFFTError_check(hipfftSetStream(subgrid_plans[i], streams[i]));

    
  }

  //Our FFT plan for our final transform.
  hipfftHandle grid_plan;
  cuFFTError_check(hipfftPlan2d(&grid_plan, grid_size, grid_size, HIPFFT_Z2Z));

  //Allocate our bins and Bin in U/V
  struct vis_data *bins;

  cudaError_check(hipMallocManaged(&bins, total_chunks * sizeof(struct vis_data), hipMemAttachGlobal));
  bin_visibilities(vis_dat, bins, chunk_count_1d, wincrement, theta, grid_size, chunk_size, &wp_min, &wp_max);
  
  //Record Start
  hipEventCreate(&start);
  hipEventRecord(start,0);
  
  int fft_gs = 32;
  int fft_bs = subgrid_size/fft_gs;
  dim3 dimGrid(fft_bs,fft_bs);
  dim3 dimBlock(fft_gs,fft_gs);

  
  // int fft_gs_m = 1024;
  //int fft_bs_m = (grid_size*grid_size)/fft_gs_m;

  dim3 dimBlock_main(16,16);
  dim3 dimGrid_main(128,128);


  double3 u_rng;
  double3 v_rng;
  double3 w_rng;
  int last_wp = wp_min;
  // Lets get gridding!

  int wkern_size = wkern_dat->size_x;
  int wkern_wstep = wkern_dat->w_step;
    
  for(int chunk =0; chunk < total_chunks; ++chunk){

    int subgrid_offset = chunk * subgrid_size * subgrid_size;
    //std::cout << "Launching kernels for chunk: " << chunk << wp_min << wp_max <<"\n";
    int cx = chunk % chunk_count_1d;
    int cy = floor(chunk / chunk_count_1d);

    int x_min = cx * chunk_size - grid_size/2;
    int y_min = cy * chunk_size - grid_size/2;

    double u_min = ((double)x_min - 0.5) / theta;
    double v_min = ((double)y_min - 0.5) / theta;
    double u_max = u_min + chunk_size / theta;
    double v_max = v_min + chunk_size / theta;

    double u_mid = (double)(x_min + chunk_size / 2) / theta;
    double v_mid = (double)(y_min + chunk_size / 2) / theta;

    u_rng = {u_min, u_max, u_mid};
    v_rng = {v_min, v_max, v_mid};

    cudaError_check(hipMemsetAsync(subgrids+subgrid_offset, 0, subgrid_mem_size, streams[chunk]));
    cudaError_check(hipMemsetAsync(subimgs+subgrid_offset, 0, subgrid_mem_size, streams[chunk]));

    
    
    for(int wp = wp_min; wp<=wp_max; ++wp){
      //std::cout << "WP: " << wp << "\n";
      double w_mid = (double)wp * wincrement;
      double w_min = ((double)wp - 0.5) * wincrement;
      double w_max = ((double)wp + 0.5) * wincrement;

      w_rng = {w_min, w_max, w_mid};
      
      scatter_grid_kernel <<< 1, 64, 0, streams[chunk] >>>
	(vis_dat, wkern_dat, subgrids+subgrid_offset, wkern_size,
	 subgrid_size, wkern_wstep, theta,
	 u_mid, v_mid, w_mid, u_rng, v_rng, w_rng);
      cuFFTError_check(hipfftExecZ2Z(subgrid_plans[chunk], subgrids+subgrid_offset, subgrids+subgrid_offset, HIPFFT_BACKWARD));
      fresnel_subimg_mul <<< dimGrid, dimBlock, 0, streams[chunk] >>> (subgrids+subgrid_offset, wtransfer, subimgs+subgrid_offset, subgrid_size, last_wp - wp);
      cudaError_check(hipMemsetAsync(subgrids+subgrid_offset, 0.0, subgrid_mem_size, streams[chunk]));
      last_wp = wp;
    }
       
    w0_transfer_kernel <<< dimGrid , dimBlock, 0, streams[chunk] >>> (subimgs+subgrid_offset, wtransfer, last_wp, subgrid_size);
    cuFFTError_check(hipfftExecZ2Z(subgrid_plans[chunk], subimgs+subgrid_offset, subimgs+subgrid_offset, HIPFFT_FORWARD)); //This can be sped up with a batched strided FFT. Future optimisation...
    

  }
  cudaError_check(hipDeviceSynchronize());
  add_subs2main_kernel <<< dimGrid_main, dimBlock_main >>> (grid_dev, subimgs, grid_size, subgrid_size,
  					  subgrid_margin, chunk_count_1d, chunk_size);
  fft_shift_kernel <<< dimGrid_main, dimBlock_main >>> (grid_dev, grid_size);
  cuFFTError_check(hipfftExecZ2Z(grid_plan, grid_dev, grid_dev, HIPFFT_BACKWARD));

  fft_shift_kernel <<< dimGrid_main, dimBlock_main >>> (grid_dev, grid_size);
  

  hipEventCreate(&stop);
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime,start,stop);
  std::cout << "Scatter Gridder Elapsed Time: " << elapsedTime/1000.0 << " seconds\n";
  
  //Transfer back to host.
  cudaError_check(hipMemcpy(grid_host, grid_dev, total_gs * sizeof(hipDoubleComplex),
  			     hipMemcpyDeviceToHost));
  
  
  //Write Image to disk on host.
  double *row = (double *) malloc(sizeof(double)*grid_size);
  std::ofstream image_f ("image.out", std::ofstream::out | std::ofstream::binary);
  std::cout << "Writing Image to File... \n";

  //  fft_shift(grid_host,grid_size);
  for(int i = 0; i < grid_size; ++i){

    for(int j = 0; j< grid_size; ++j){

      row[j] = hipCreal(grid_host[i*grid_size + j]);
    }
    image_f.write((char*)row, sizeof(double) * grid_size);
  }

  image_f.close();
  
  //Check it actually ran...
  hipError_t err = hipGetLastError();
  std::cout << "Error: " << hipGetErrorString(err) << "\n";
  
  cudaError_check(hipDeviceReset());
  
  
  return error;

}

//W-Towers Wrapper.
__host__ hipError_t wtowers_CUDA_flat(const char* visfile, const char* wkernfile, int grid_size,
			   double theta,  double lambda, double bl_min, double bl_max,
				  int subgrid_size, int subgrid_margin, double wincrement){

  //API Variables
  hipError_t error = (hipError_t)0;
  
 
  //For Benchmarking.
  hipEvent_t start, stop;
  float elapsedTime;

  // Load visibility and w-kernel data from HDF5 files.
  struct vis_data *vis_dat;
  struct flat_vis_data *flat_vis_dat, *vis_bins;
  struct w_kernel_data *wkern_dat;

  cudaError_check(hipMallocManaged((void **)&vis_dat, sizeof(struct vis_data), hipMemAttachGlobal));
  cudaError_check(hipMallocManaged((void **)&wkern_dat, sizeof(struct w_kernel_data), hipMemAttachGlobal));

  int error_hdf5;
  error_hdf5 = load_vis(visfile,vis_dat,bl_min,bl_max);
  if (error_hdf5) {
    std::cout << "Failed to Load Visibilities \n";
    return error;
  }
  error_hdf5 = load_wkern_CUDA(wkernfile, theta, wkern_dat);
  if (error_hdf5) {
    std::cout << "Failed to Load W-Kernels \n";
    return error;
  }

  cudaError_check(hipHostMalloc((void**)&flat_vis_dat, sizeof(struct flat_vis_data)));
  cudaError_check(hipMallocManaged((void**)&vis_bins, sizeof(struct flat_vis_data)));
  flatten_visibilities_CUDA(vis_dat, flat_vis_dat);
  bin_flat_visibilities(vis_bins, flat_vis_dat, 256);
  // Work out our minimum and maximum w-planes.

  double vis_w_min = 0, vis_w_max = 0;
  for (int bl = 0; bl < vis_dat->bl_count; ++bl){
    double w_min = lambda_min(&vis_dat->bl[bl], vis_dat->bl[bl].w_min);
    double w_max = lambda_max(&vis_dat->bl[bl], vis_dat->bl[bl].w_max);
    if (w_min < vis_w_min) { vis_w_min = w_min; }
    if (w_max > vis_w_max) { vis_w_max = w_max; }
  }
  int wp_min = (int) floor(vis_w_min / wincrement + 0.5);
  int wp_max = (int) floor(vis_w_max / wincrement + 0.5);
  std::cout << "Our W-Plane Min/Max: " << wp_min << " " << wp_max << "\n";


  
  //Allocate our main grid.
  
  int total_gs = grid_size * grid_size;
  
  hipDoubleComplex *grid_dev, *grid_host;
  cudaError_check(hipMalloc((void **)&grid_dev, total_gs * sizeof(hipDoubleComplex)));
  cudaError_check(hipHostMalloc((void **)&grid_host, total_gs * sizeof(hipDoubleComplex)));





  //Create the fresnel interference pattern for the W-Dimension
  //Can make this a kernel.
  //See Tim Cornwells paper on W-Projection for more information.
  int subgrid_mem_size = sizeof(hipDoubleComplex) * subgrid_size * subgrid_size;  
  hipDoubleComplex *wtransfer;
  cudaError_check(hipMallocManaged((void **)&wtransfer, subgrid_mem_size, hipMemAttachGlobal));
  std::cout << "Generating Fresnel Pattern... \n";
  int x,y;
  for (y=0; y < subgrid_size; ++y){
    for (x=0; x < subgrid_size; ++x){
      double l = theta * (double)(x - subgrid_size / 2) / subgrid_size;
      double m = theta * (double)(y - subgrid_size / 2) / subgrid_size;
      double ph = wincrement * (1 - sqrt(1 - l*l - m*m));
      hipDoubleComplex wtrans = make_hipDoubleComplex(0.0, 2 * M_PI * ph);
      wtransfer[y * subgrid_size + x] = cu_cexp_d(wtrans);
    }
  }
  fft_shift(wtransfer, subgrid_size);

  //Allocate subgrids/subimgs on the GPU
  
  assert( grid_size % subgrid_size == 0);
  
  int chunk_size = subgrid_size - subgrid_margin;
  int chunk_count_1d = grid_size / chunk_size + 1;
  int total_chunks = chunk_count_1d * chunk_count_1d;

  //Allocate all our subgrids/subimgs contiguously.
  hipDoubleComplex *subgrids, *subimgs;
  
  cudaError_check(hipMallocManaged((void **)&subgrids,
				    total_chunks * subgrid_mem_size  * sizeof(hipDoubleComplex),
				    hipMemAttachGlobal));
  cudaError_check(hipMallocManaged((void **)&subimgs,
				    total_chunks * subgrid_mem_size * sizeof(hipDoubleComplex),
				    hipMemAttachGlobal));

  //  cudaError_check(hipMemset(subimgs, 1.2, total_chunks * subgrid_mem_size * sizeof(hipDoubleComplex)));
  //Create streams for each tower and allocate our chunks in unified memory.
  //Also set our FFT plans while we are here.
  //Initialise cublas handle. We use cublas to multiply our fresnel phase screen.

  hipStream_t *streams = (hipStream_t *) malloc(total_chunks * sizeof(hipStream_t));
  hipfftHandle *subgrid_plans = (hipfftHandle *) malloc(total_chunks * sizeof(hipfftHandle));
  
  for(int i = 0; i < total_chunks; ++i){

    //Create stream.
    cudaError_check(hipStreamCreate(&streams[i]));

    //Assign FFT Plan to each stream
    cuFFTError_check(hipfftPlan2d(&subgrid_plans[i],subgrid_size,subgrid_size,HIPFFT_Z2Z));
    cuFFTError_check(hipfftSetStream(subgrid_plans[i], streams[i]));

    
  }

  //Our FFT plan for our final transform.
  hipfftHandle grid_plan;
  cuFFTError_check(hipfftPlan2d(&grid_plan, grid_size, grid_size, HIPFFT_Z2Z));

  //Allocate our bins and Bin in U/V
  //struct vis_data *bins;

  //cudaError_check(hipMallocManaged(&bins, total_chunks * sizeof(struct vis_data), hipMemAttachGlobal));
  //bin_visibilities(vis_dat, bins, chunk_count_1d, wincrement, theta, grid_size, chunk_size, &wp_min, &wp_max);
  
  //Record Start
  hipEventCreate(&start);
  hipEventRecord(start,0);
  
  int fft_gs = 32;
  int fft_bs = subgrid_size/fft_gs;
  dim3 dimGrid(fft_bs,fft_bs);
  dim3 dimBlock(fft_gs,fft_gs);

  
  // int fft_gs_m = 1024;
  //int fft_bs_m = (grid_size*grid_size)/fft_gs_m;

  dim3 dimBlock_main(16,16);
  dim3 dimGrid_main(128,128);


  double3 u_rng;
  double3 v_rng;
  double3 w_rng;
  int last_wp = wp_min;
  // Lets get gridding!

  int wkern_size = wkern_dat->size_x;
  int wkern_wstep = wkern_dat->w_step;
    
  for(int chunk =0; chunk < total_chunks; ++chunk){

    int subgrid_offset = chunk * subgrid_size * subgrid_size;
    //std::cout << "Launching kernels for chunk: " << chunk << wp_min << wp_max <<"\n";
    int cx = chunk % chunk_count_1d;
    int cy = floor(chunk / chunk_count_1d);

    int x_min = cx * chunk_size - grid_size/2;
    int y_min = cy * chunk_size - grid_size/2;

    double u_min = ((double)x_min - 0.5) / theta;
    double v_min = ((double)y_min - 0.5) / theta;
    double u_max = u_min + chunk_size / theta;
    double v_max = v_min + chunk_size / theta;

    double u_mid = (double)(x_min + chunk_size / 2) / theta;
    double v_mid = (double)(y_min + chunk_size / 2) / theta;

    u_rng = {u_min, u_max, u_mid};
    v_rng = {v_min, v_max, v_mid};

    cudaError_check(hipMemsetAsync(subgrids+subgrid_offset, 0, subgrid_mem_size, streams[chunk]));
    cudaError_check(hipMemsetAsync(subimgs+subgrid_offset, 0, subgrid_mem_size, streams[chunk]));

    
    
    for(int wp = wp_min; wp<=wp_max; ++wp){
      //std::cout << "WP: " << wp << "\n";
      double w_mid = (double)wp * wincrement;
      double w_min = ((double)wp - 0.5) * wincrement;
      double w_max = ((double)wp + 0.5) * wincrement;

      w_rng = {w_min, w_max, w_mid};
      
      scatter_grid_kernel_flat <<< 256, 64, 0, streams[chunk] >>>
	(vis_bins, wkern_dat, subgrids+subgrid_offset, wkern_size,
	 subgrid_size, wkern_wstep, theta,
	 u_mid, v_mid, w_mid, u_rng, v_rng, w_rng);
      cuFFTError_check(hipfftExecZ2Z(subgrid_plans[chunk], subgrids+subgrid_offset, subgrids+subgrid_offset, HIPFFT_BACKWARD));
      fresnel_subimg_mul <<< dimGrid, dimBlock, 0, streams[chunk] >>> (subgrids+subgrid_offset, wtransfer, subimgs+subgrid_offset, subgrid_size, last_wp - wp);
      cudaError_check(hipMemsetAsync(subgrids+subgrid_offset, 0.0, subgrid_mem_size, streams[chunk]));
      last_wp = wp;
    }
       
    w0_transfer_kernel <<< dimGrid , dimBlock, 0, streams[chunk] >>> (subimgs+subgrid_offset, wtransfer, last_wp, subgrid_size);
    cuFFTError_check(hipfftExecZ2Z(subgrid_plans[chunk], subimgs+subgrid_offset, subimgs+subgrid_offset, HIPFFT_FORWARD)); //This can be sped up with a batched strided FFT. Future optimisation...
    

  }
  cudaError_check(hipDeviceSynchronize());
  add_subs2main_kernel <<< dimGrid_main, dimBlock_main >>> (grid_dev, subimgs, grid_size, subgrid_size,
  					  subgrid_margin, chunk_count_1d, chunk_size);
  fft_shift_kernel <<< dimGrid_main, dimBlock_main >>> (grid_dev, grid_size);
  cuFFTError_check(hipfftExecZ2Z(grid_plan, grid_dev, grid_dev, HIPFFT_BACKWARD));

  fft_shift_kernel <<< dimGrid_main, dimBlock_main >>> (grid_dev, grid_size);
  

  hipEventCreate(&stop);
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime,start,stop);
  std::cout << "Scatter Gridder Elapsed Time: " << elapsedTime/1000.0 << " seconds\n";
  
  //Transfer back to host.
  cudaError_check(hipMemcpy(grid_host, grid_dev, total_gs * sizeof(hipDoubleComplex),
  			     hipMemcpyDeviceToHost));
  
  
  //Write Image to disk on host.
  double *row = (double *) malloc(sizeof(double)*grid_size);
  std::ofstream image_f ("image.out", std::ofstream::out | std::ofstream::binary);
  std::cout << "Writing Image to File... \n";

  //  fft_shift(grid_host,grid_size);
  for(int i = 0; i < grid_size; ++i){

    for(int j = 0; j< grid_size; ++j){

      row[j] = hipCreal(grid_host[i*grid_size + j]);
    }
    image_f.write((char*)row, sizeof(double) * grid_size);
  }

  image_f.close();
  
  //Check it actually ran...
  hipError_t err = hipGetLastError();
  std::cout << "Error: " << hipGetErrorString(err) << "\n";
  
  cudaError_check(hipDeviceReset());
  
  
  return error;

}



// Pure W-Projection on a Hierarchical Dataset. (AoS)
__host__ hipError_t wprojection_CUDA(const char* visfile, const char* wkernfile, int grid_size,
				      double theta,  double lambda, double bl_min, double bl_max,
				      int threads_per_block){

  //For Benchmarking.
  
  hipError_t error = (hipError_t)0; //Initialise as CUDA_Success
  hipEvent_t start, stop;
  float elapsedTime;

  // Load visibility and w-kernel data from HDF5 files.
  
  struct vis_data *vis_dat;
  struct w_kernel_data *wkern_dat;

  cudaError_check(hipMallocManaged((void **)&vis_dat, sizeof(struct vis_data), hipMemAttachGlobal));
  cudaError_check(hipMallocManaged((void **)&wkern_dat, sizeof(struct w_kernel_data), hipMemAttachGlobal));

  int error_hdf5;
  error_hdf5 = load_vis_CUDA(visfile,vis_dat,bl_min,bl_max);
  if (error_hdf5) {
    std::cout << "Failed to Load Visibilities \n";
    return error;
  }
  error_hdf5 = load_wkern_CUDA(wkernfile, theta, wkern_dat);
  if (error_hdf5) {
    std::cout << "Failed to Load W-Kernels \n";
    return error;
  }

  

  //Allocate our main grid.
  
  int total_gs = grid_size * grid_size;
  
  hipDoubleComplex *grid_dev, *grid_host;
  cudaError_check(hipMalloc((void **)&grid_dev, total_gs * sizeof(hipDoubleComplex)));
  cudaError_check(hipHostMalloc((void **)&grid_host, total_gs * sizeof(hipDoubleComplex)));


  
  //int blocks = total_gs / 256;

  //Weight visibilities

  weight((unsigned int *)grid_host, grid_size, theta, vis_dat);
    
  std::cout << "Inititalise scatter gridder... \n";

  hipEventCreate(&start);
  hipEventRecord(start, 0);

  double3 u_rng = {-10e100,10e100,0};
  double3 v_rng = {-10e100,10e100,0};
  double3 w_rng = {-10e100,10e100,0};

  //scatter_grid_kernel_flat <<< 16, 32 >>> (flat_vis_dat, wkern_dat, grid_dev, wkern_dat->size_x,
  //					  grid_size, grid_size, wkern_dat->w_step, theta, 0, 0, 0);
  //
  scatter_grid_kernel <<< 16 , 32 >>> (vis_dat,wkern_dat, grid_dev, wkern_dat->size_x,
				       grid_size,wkern_dat->w_step, theta, 0, 0, 0, u_rng, v_rng, w_rng);
  hipEventCreate(&stop);
  hipEventRecord(stop, 0);

  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime,start,stop);

  std::cout << "Scatter Gridder Elapsed Time: " << elapsedTime/1000.0 << " seconds\n";

  //Shift our grid to the right position for the FFT.
  int fft_gs = 32;
  int fft_bs = grid_size / fft_gs;
  
  dim3 dimBlock(fft_bs,fft_bs);
  dim3 dimGrid(fft_gs,fft_gs);
  fft_shift_kernel <<< dimBlock, dimGrid >>> (grid_dev, grid_size);
  

    //Transfer back to host.
  cudaError_check(hipMemcpy(grid_host, grid_dev, total_gs * sizeof(hipDoubleComplex),
			     hipMemcpyDeviceToHost));



//Write Image to disk on host.

  std::ofstream image_pref ("pre_fft.out", std::ofstream::out | std::ofstream::binary);
  std::cout << "Writing Image to File... \n";

  double *row;
  cudaError_check(hipHostMalloc(&row, grid_size * sizeof(double)));

      
  for(int i = 0; i < grid_size; ++i){

    for(int j = 0; j< grid_size; ++j){
      
      row[j] = hipCreal(grid_host[i*grid_size + j]);
    }
    image_pref.write((char*)row, sizeof(double) * grid_size);
  }
  image_pref.close();


  
  //fft_shift(grid_host, grid_size);
  make_hermitian(grid_host, grid_size);


  
  cudaError_check(hipMemcpy(grid_dev, grid_host, total_gs * sizeof(hipDoubleComplex),
			     hipMemcpyHostToDevice));
  
  std::cout << "Executing iFFT back to Image Space... \n";
  
  hipfftHandle fft_plan;
  cuFFTError_check(hipfftPlan2d(&fft_plan,grid_size,grid_size,HIPFFT_Z2Z));
  cuFFTError_check(hipfftExecZ2Z(fft_plan, grid_dev, grid_dev, HIPFFT_BACKWARD));
  fft_shift_kernel <<< dimBlock, dimGrid >>> (grid_dev, grid_size);
  //Transfer back to host.
  cudaError_check(hipMemcpy(grid_host, grid_dev, total_gs * sizeof(hipDoubleComplex),
			     hipMemcpyDeviceToHost));


  //Write Image to disk on host.

  std::ofstream image_f ("image.out", std::ofstream::out | std::ofstream::binary);
  std::cout << "Writing Image to File... \n";

  //double *row;
  //cudaError_check(hipHostMalloc(&row, grid_size * sizeof(double)));
    for(int i = 0; i < grid_size; ++i){

    for(int j = 0; j< grid_size; ++j){

      row[j] = hipCreal(grid_host[i*grid_size + j]);
    }
    image_f.write((char*)row, sizeof(double) * grid_size);
  }

  image_f.close();

  //Check it actually ran...
  hipError_t err = hipGetLastError();
  

  std::cout << "Error: " << hipGetErrorString(err) << "\n";
  return err;
}

// W-Project on flat SoA Dataset.
__host__ hipError_t wprojection_CUDA_flat(const char* visfile, const char* wkernfile, int grid_size,
				      double theta,  double lambda, double bl_min, double bl_max,
				      int threads_per_block){

  hipDeviceSetCacheConfig(hipFuncCachePreferL1);
  //For Benchmarking.
  
  hipError_t error = (hipError_t)0;
  hipEvent_t start, stop;
  float elapsedTime;

  // Load visibility and w-kernel data from HDF5 files.
  
  struct vis_data *vis_dat = (struct vis_data*)malloc(sizeof(struct vis_data));

  vis_dat->antenna_count = 0;
  
  struct w_kernel_data *wkern_dat;

  //hipHostMalloc((void **)&vis_dat, sizeof(struct vis_data));
  //cudaError_check(hipHostMalloc((void **)&vis_dat, sizeof(struct vis_data)));
  cudaError_check(hipMallocManaged((void **)&wkern_dat, sizeof(struct w_kernel_data), hipMemAttachGlobal));

  int error_hdf5;
  error_hdf5 = load_vis(visfile,vis_dat,bl_min,bl_max);
  if (error_hdf5) {
    std::cout << "Failed to Load Visibilities \n";
    return error;
  }
  error_hdf5 = load_wkern_CUDA(wkernfile, theta, wkern_dat);
  if (error_hdf5) {
    std::cout << "Failed to Load W-Kernels \n";
    return error;
  }

  //Allocate our main grid.
  
  int total_gs = grid_size * grid_size;
  
  hipDoubleComplex *grid_dev, *grid_host;
  cudaError_check(hipMalloc((void **)&grid_dev, total_gs * sizeof(hipDoubleComplex)));
  cudaError_check(hipHostMalloc((void **)&grid_host, total_gs * sizeof(hipDoubleComplex)));

  //Make sure our grids are all zero.
  
  cudaError_check(hipMemset(grid_dev, 0, total_gs * sizeof(hipDoubleComplex)));
  cudaError_check(hipMemset(grid_host, 0, total_gs * sizeof(hipDoubleComplex)));

  
  struct flat_vis_data *flat_vis_dat;
  cudaError_check(hipHostMalloc((void**)&flat_vis_dat, sizeof(struct flat_vis_data)));

  //Flatten the visibilities and weight them.
  flatten_visibilities_CUDA(vis_dat,flat_vis_dat);
  weight_flat((unsigned int *)grid_host, grid_size, theta, flat_vis_dat);


  
  //Now bin them per block.
  struct flat_vis_data *vis_bins;
  cudaError_check(hipMallocManaged((void**)&vis_bins, sizeof(struct flat_vis_data) * 1024, hipMemAttachGlobal));

  bin_flat_visibilities(vis_bins, flat_vis_dat, 1024);

  double3 u_rng{-1e300,1e300,0};
  double3 v_rng{-1e300,1e300,0};
  double3 w_rng{-1e300,1e300,0};
  
  //Get scattering..
  std::cout << "Inititalise scatter gridder... \n";
  hipEventCreate(&start);
  hipEventRecord(start, 0);

  scatter_grid_kernel_flat <<< 1024, 256 >>> (vis_bins, wkern_dat, grid_dev, wkern_dat->size_x,
					      grid_size, wkern_dat->w_step, theta, 0, 0, 0,
					      u_rng, v_rng, w_rng);
  
  hipEventCreate(&stop);
  hipEventRecord(stop, 0);

  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime,start,stop);

  std::cout << "Scatter Gridder Elapsed Time: " << elapsedTime/1000.0 << " seconds\n";

  
  cudaError_check(hipDeviceSynchronize());

  //Shift our grid to the right position for the FFT.
  int fft_gs = 32;
  int fft_bs = grid_size / fft_gs;
  
  dim3 dimBlock(fft_bs,fft_bs);
  dim3 dimGrid(fft_gs,fft_gs);

  //Transfer back to host.
  cudaError_check(hipMemcpy(grid_host, grid_dev, total_gs * sizeof(hipDoubleComplex),
			     hipMemcpyDeviceToHost));



  //Write Grid to disk on host.

  std::ofstream image_pref ("pre_fft.out", std::ofstream::out | std::ofstream::binary);
  std::cout << "Writing Image to File... \n";
  
  double *row;
  cudaError_check(hipHostMalloc(&row, grid_size * sizeof(double)));

     
  for(int i = 0; i < grid_size; ++i){

    for(int j = 0; j< grid_size; ++j){
      
      row[j] = hipCreal(grid_host[i*grid_size + j]);
    }
    image_pref.write((char*)row, sizeof(double) * grid_size);
  }
  image_pref.close();


  
  
  
  make_hermitian(grid_host, grid_size);


  
  cudaError_check(hipMemcpy(grid_dev, grid_host, total_gs * sizeof(hipDoubleComplex),
  			     hipMemcpyHostToDevice));
  fft_shift_kernel <<< dimBlock, dimGrid >>> (grid_dev, grid_size);
  std::cout << "Executing iFFT back to Image Space... \n";
  
  hipfftHandle fft_plan;
  cuFFTError_check(hipfftPlan2d(&fft_plan,grid_size,grid_size,HIPFFT_Z2Z));
  cuFFTError_check(hipfftExecZ2Z(fft_plan, grid_dev, grid_dev, HIPFFT_BACKWARD));
  fft_shift_kernel <<< dimBlock, dimGrid >>> (grid_dev, grid_size);
  //Transfer back to host.
  cudaError_check(hipMemcpy(grid_host, grid_dev, total_gs * sizeof(hipDoubleComplex),
			     hipMemcpyDeviceToHost));


  //Write Image to disk on host.

  std::ofstream image_f ("image.out", std::ofstream::out | std::ofstream::binary);
  std::cout << "Writing Image to File... \n";

  //  fft_shift(grid_host,grid_size);
  for(int i = 0; i < grid_size; ++i){

    for(int j = 0; j< grid_size; ++j){

      row[j] = hipCreal(grid_host[i*grid_size + j]);
    }
    image_f.write((char*)row, sizeof(double) * grid_size);
  }

  image_f.close();

  //Check it actually ran...
  hipError_t err = hipGetLastError();
  std::cout << "Error: " << hipGetErrorString(err) << "\n";

  cudaError_check(hipDeviceReset());
  
  return err;
} 
