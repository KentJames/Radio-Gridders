#include "hip/hip_runtime.h"
//C++ Includes
#include <iostream>
#include <fstream>
#include <cstdlib>
#include <cassert>

//CUDA Includes
#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>
#include <hipblas.h>
#include "hip/hip_runtime.h"
#include "math.h"
#include "hip/hip_runtime_api.h"

//Our Include
#include "wtowers_common.h"


/*****************************
      CUDA Error Checker
******************************/
 
#define cudaError_check(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){

  if (code != hipSuccess){
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

#define cuFFTError_check(ans) { cufftAssert((ans), __FILE__, __LINE__); }
inline void cufftAssert(hipfftResult code, const char *file, int line, bool abort=true){

  if (code != HIPFFT_SUCCESS){
    fprintf(stderr,"cufftAssert: %d %s %d\n", code, file, line);
    if (abort) exit(code);
  }
}

/*****************************
        Device Functions
 *****************************/

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600


#else //Pre-pascal devices.

__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

#endif

__host__ __device__ inline hipDoubleComplex cu_cexp_d (hipDoubleComplex z){

  hipDoubleComplex res;
  double t = exp (z.x);
  sincos (z.y, &res.y, &res.x);
  res.x *= t;
  res.y *= t;
  return res;

}

__host__ __device__ inline static double uvw_lambda(struct bl_data *bl_data,
				  int time, int freq, int uvw) {
    return bl_data->uvw[3*time+uvw] * bl_data->freq[freq] / c;
    
  }

__host__ __device__ inline static void frac_coord_flat(int grid_size, int kernel_size, int oversample,
                              double theta,
                              struct flat_vis_data *vis,
                              int i,
                              double d_u, double d_v,
                              int *grid_offset, int *sub_offset) {
#ifdef ASSUME_UVW_0
    double x = 0, y = 0;
#else
    double x = theta * (vis->u[i] - d_u);
    double y = theta * (vis->v[i] - d_v);
#endif
    int flx = (int)floor(x + .5 / oversample);
    int fly = (int)floor(y + .5 / oversample);
    int xf = (int)floor((x - (double)flx) * oversample + .5);
    int yf = (int)floor((y - (double)fly) * oversample + .5);
    *grid_offset =
        (fly+grid_size/2-kernel_size/2)*grid_size +
        (flx+grid_size/2-kernel_size/2);
    *sub_offset = kernel_size * kernel_size * (yf*oversample + xf);
}


__host__ __device__ inline static void frac_coord(int grid_size, int kernel_size, int oversample,
                              double theta,
                              struct bl_data *bl_data,
                              int time, int freq,
                              double d_u, double d_v,
                              int *grid_offset, int *sub_offset) {
#ifdef ASSUME_UVW_0
    double x = 0, y = 0;
#else
    double x = theta * (uvw_lambda(bl_data, time, freq, 0) - d_u);
    double y = theta * (uvw_lambda(bl_data, time, freq, 1) - d_v);
#endif
    int flx = (int)floor(x + .5 / oversample);
    int fly = (int)floor(y + .5 / oversample);
    int xf = (int)floor((x - (double)flx) * oversample + .5);
    int yf = (int)floor((y - (double)fly) * oversample + .5);
    *grid_offset =
        (fly+grid_size/2-kernel_size/2)*grid_size +
        (flx+grid_size/2-kernel_size/2);
    *sub_offset = kernel_size * kernel_size * (yf*oversample + xf);
}


//From Kyrills implementation in SKA/RC
__device__ inline void scatter_grid_add(hipDoubleComplex *uvgrid, int grid_size, int grid_pitch,
					int grid_point_u, int grid_point_v, hipDoubleComplex sum){

  
  // Atomically add to grid. This is the bottleneck of this kernel.
  if (grid_point_u < 0 || grid_point_u >= grid_size ||
      grid_point_v < 0 || grid_point_v >= grid_size)
    return;

  // Bottom half? Mirror
  //if (grid_point_u >= grid_size / 2) {
  //  grid_point_v = grid_size - grid_point_v - 1;
  //  grid_point_u = grid_size - grid_point_u - 1;
  //}

  // Add to grid. This is the bottleneck of the entire kernel
  atomicAdd(&uvgrid[grid_point_u + grid_pitch*grid_point_v].x, sum.x);
  atomicAdd(&uvgrid[grid_point_u + grid_pitch*grid_point_v].y, sum.y);

}


__device__ inline void scatter_grid_point_flat(
					  struct flat_vis_data *vis, // Our bins of UV Data
					  hipDoubleComplex *uvgrid, // Our main UV Grid
					  struct w_kernel_data *wkern, //Our W-Kernel
					  int max_supp, // Max size of W-Kernel
					  int myU, //Our assigned u/v points.
					  int myV, // ^^^
					  double wstep, // W-Increment 
					  int subgrid_size, //The size of our w-towers subgrid.
					  int subgrid_pitch, // Not too sure about ths one
					  double theta, // Field of View Size
					  int offset_u, // Offset from top left of main grid to t.l of subgrid.
					  int offset_v, // ^^^^
					  int offset_w
					  ){ 

  int grid_point_u = myU, grid_point_v = myV;
  hipDoubleComplex sum  = make_hipDoubleComplex(0.0,0.0);

  short supp = short(wkern->size_x);
  
  //  for (int i = 0; i < visibilities; i++) {
  int vi;
  for (vi = 0; vi < vis->number_of_vis; ++vi){
    

    
    //double u = vis->u[vi];
    //double v = vis->v[vi];
    double w = vis->w[vi];
    int w_plane = fabs((w - wkern->w_min) / (wkern->w_step + .5));
    int grid_offset, sub_offset;
    frac_coord_flat(subgrid_size, wkern->size_x, wkern->oversampling,
		    theta, vis, vi, offset_u, offset_v, &grid_offset, &sub_offset);
    int u = grid_offset % subgrid_size; 
    int v = grid_offset / subgrid_size;

    // Determine convolution point. This is basically just an
    // optimised way to calculate
    //   myConvU = (myU - u) % max_supp
    //   myConvV = (myV - v) % max_supp
    //	int2 xy = getcoords_xy(u,v,subgrid_size,theta,max_supp);
    int myConvU = (u - myU) % max_supp;
    int myConvV = (v - myV) % max_supp;
    if (myConvU < 0) myConvU += max_supp;
    if (myConvV < 0) myConvV += max_supp;

    // Determine grid point. Because of the above we know here that
    //   myGridU % max_supp = myU
    //   myGridV % max_supp = myV
    int myGridU = u + myConvU
      , myGridV = v + myConvV;

    // Grid point changed?
    if (myGridU != grid_point_u || myGridV != grid_point_v) {
      // Atomically add to grid. This is the bottleneck of this kernel.
      scatter_grid_add(uvgrid, subgrid_size, subgrid_pitch, grid_point_u, grid_point_v, sum);
      // Switch to new point
      sum = make_hipDoubleComplex(0.0, 0.0);
      grid_point_u = myGridU;
      grid_point_v = myGridV;
    }
    //TODO: Re-do the w-kernel/gcf for our data.
    //	hipDoubleComplex px;
    hipDoubleComplex px = *(hipDoubleComplex*)&wkern->kern_by_w[w_plane].data[sub_offset + myConvU * supp + myConvV];	
    // Sum up
    hipDoubleComplex vi_v = *(hipDoubleComplex*)&vis->vis[vi];
    sum = hipCfma(px, vi_v, sum);
      
    
  }

  // Add remaining sum to grid
  scatter_grid_add(uvgrid, subgrid_size, subgrid_pitch, grid_point_u, grid_point_v, sum);

}



//From Kyrills Implementation in SKA/RC. Modified to suit our data format.
//Assumes pre-binned (in u/v) data
__device__ inline void scatter_grid_point(
					  struct bl_data **bin, // Our bins of UV Data
					  int bl_count, // Number of baselines.
					  hipDoubleComplex *uvgrid, // Our main UV Grid
					  struct w_kernel_data *wkern, //Our W-Kernel
					  int max_supp, // Max size of W-Kernel
					  int myU, //Our assigned u/v points.
					  int myV, // ^^^
					  double wstep, // W-Increment 
					  int subgrid_size, //The size of our w-towers subgrid.
					  int subgrid_pitch, // Not too sure about ths one
					  double theta, // Field of View Size
					  int offset_u, // Offset from top left of main grid to t.l of subgrid.
					  int offset_v, // ^^^^
					  int offset_w
					  ){ 

  int grid_point_u = myU, grid_point_v = myV;
  hipDoubleComplex sum  = make_hipDoubleComplex(0.0,0.0);

  short supp = short(wkern->size_x);
  
  //  for (int i = 0; i < visibilities; i++) {
  int bl, time, freq;
  for (bl = 0; bl < bl_count; ++bl){
    struct bl_data *bl_d = *bin+bl;
    for (time = 0; time < bl_d->time_count; ++time){
      for(freq = 0; freq < bl_d->freq_count; ++freq){
	// Load pre-calculated positions
	//int u = uvo[i].u, v = uvo[i].v;
	//	int u = (int)uvw_lambda(bl_d, time, freq, 0);
	//int v = (int)uvw_lambda(bl_d, time, freq, 1);
	double w = uvw_lambda(bl_d, time, freq, 2) - offset_w;
	int w_plane = fabs((w - wkern->w_min) / (wkern->w_step + .5));
	int grid_offset, sub_offset;
	frac_coord(subgrid_size, wkern->size_x, wkern->oversampling,
		   theta, bl_d, time, freq, offset_u, offset_v, &grid_offset, &sub_offset);
	int u = grid_offset % subgrid_size; 
	int v = grid_offset / subgrid_size;

	// Determine convolution point. This is basically just an
	// optimised way to calculate
	//   myConvU = (myU - u) % max_supp
	//   myConvV = (myV - v) % max_supp
	//	int2 xy = getcoords_xy(u,v,subgrid_size,theta,max_supp);
	int myConvU = (u - myU) % max_supp;
	int myConvV = (v - myV) % max_supp;
	if (myConvU < 0) myConvU += max_supp;
	if (myConvV < 0) myConvV += max_supp;

	// Determine grid point. Because of the above we know here that
	//   myGridU % max_supp = myU
	//   myGridV % max_supp = myV
	int myGridU = u + myConvU
	  , myGridV = v + myConvV;

	// Grid point changed?
	if (myGridU != grid_point_u || myGridV != grid_point_v) {
	  // Atomically add to grid. This is the bottleneck of this kernel.
	  scatter_grid_add(uvgrid, subgrid_size, subgrid_pitch, grid_point_u, grid_point_v, sum);
	  // Switch to new point
	  sum = make_hipDoubleComplex(0.0, 0.0);
	  grid_point_u = myGridU;
	  grid_point_v = myGridV;
	  }
	//TODO: Re-do the w-kernel/gcf for our data.
	//	hipDoubleComplex px;
	hipDoubleComplex px = *(hipDoubleComplex*)&wkern->kern_by_w[w_plane].data[sub_offset + myConvU * supp + myConvV];	
	// Sum up
	hipDoubleComplex vi = *(hipDoubleComplex*)&bl_d->vis[time*bl_d->freq_count+freq];
	sum = hipCfma(px, vi, sum);
      }
    }
  }

  // Add remaining sum to grid
  scatter_grid_add(uvgrid, subgrid_size, subgrid_pitch, grid_point_u, grid_point_v, sum);

}


/******************************
            Kernels
*******************************/


//Multiplies our (inverse transformeD!) grid by the fresnel pattern and
//then adds it to our subimg. Uses cuBLAS
inline void fresnel_blas_mmul(hipblasHandle_t &handle,
		       hipDoubleComplex *subgrid,
		       hipDoubleComplex *fresnel,
		       hipDoubleComplex *subimg,
		       int n){

  hipDoubleComplex alf = make_hipDoubleComplex(1.0,1.0);
  hipDoubleComplex bet = make_hipDoubleComplex(0.0,0.0);

  hipDoubleComplex *alpha = &alf;
  hipDoubleComplex *beta = &bet;

  //Complex->Complex cublas matrix multiplication.
  hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, alpha,
	      subgrid, n, fresnel, n, beta, subgrid, n);

  hipblasZaxpy(handle, n*n, beta, subgrid, 1, subimg, 1);
  

}

//Shifts a 2D grid to be in the right place for an FFT. 
__global__ void fft_shift_kernel(hipDoubleComplex *grid, int size){

  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if(x<size/2 && y <size){

    int ix0 = y * size + x;
    int ix1 = (ix0 + (size + 1) * (size/2)) % (size*size);

    hipDoubleComplex temp = grid[ix0];
    grid[ix0] = grid[ix1];
    grid[ix1] = temp;


  }

}

//This is our Romein-style scatter gridder. Works on flat visibility data.
__global__ void scatter_grid_kernel_flat(
				    struct flat_vis_data *vis, // No. of visibilities
				    struct w_kernel_data *wkern, // No. of wkernels
				    hipDoubleComplex *uvgrid, //Our UV-Grid
				    int max_support, //  Convolution size
				    int subgrid_size, // Subgrid size
				    int subgrid_pitch, // Subgrid pitch (what is this?)
				    double wstep, // W-Increment
				    double theta, // Field of View
				    int offset_u, // Top left offset from top left main grid
				    int offset_v, // ^^^^
				    int offset_w // W Offset
				    ){

  //Assign some visibilities to grid;

  

  
  for(int i = threadIdx.x; i < max_support * max_support; i += blockDim.x){
    //  int i = threadIdx.x + blockIdx.x * blockDim.x;
    int myU = i % max_support;
    int myV = i / max_support;

    scatter_grid_point_flat(vis+blockIdx.x, uvgrid, wkern, max_support, myU, myV, wstep,
		       subgrid_size, subgrid_pitch, theta, offset_u, offset_v, offset_w);
		       
  }
}


//This is our Romein-style scatter gridder. Works on hierarchical visibility data (bl->time->freq)
__global__ void scatter_grid_kernel(struct bl_data **bin, // Baseline bin
				    int bl_count, // No. of baselines
				    struct vis_data *vis, // No. of visibilities
				    struct w_kernel_data *wkern, // No. of wkernels
				    hipDoubleComplex *uvgrid, //Our UV-Grid
				    int max_support, //  Convolution size
				    int subgrid_size, // Subgrid size
				    int subgrid_pitch, // Subgrid pitch (what is this?)
				    double wstep, // W-Increment
				    double theta, // Field of View
				    int offset_u, // Top left offset from top left main grid
				    int offset_v, // ^^^^
				    int offset_w // W Offset
				    ){
  
  for(int i = threadIdx.x; i < max_support * max_support; i += blockDim.x){
    //  int i = threadIdx.x + blockIdx.x * blockDim.x;
    int myU = i % max_support;
    int myV = i / max_support;

    scatter_grid_point(bin, bl_count, uvgrid, wkern, max_support, myU, myV, wstep,
		       subgrid_size, subgrid_pitch, theta, offset_u, offset_v, offset_w);
		       
  }
}


//This kernel uses CUDA's dynamic parallelism (compute > 35) to repeatedly call kernels to
//grid and then multiply the fresnel pattern for each CHUNK. The result is atomically added to the uvgrid.
//Unfortunately this has to be host managed due to cufft being a host API.
__global__ void wtowers_kernel(hipDoubleComplex *subimg, //Subimg (Image Space)
			       hipDoubleComplex *subgrid, //Subgrid (Fourier Space)
			       hipDoubleComplex *fresnel, // The fresnel pattern
			       struct bl_data **bin, // Baseline bin for our chunk
			       int subgrid_size, //Grid size in pixels
			       double wstep, // Increment/delta between w-planes
			       double theta, // Field of View
			       int offset_u, //Offset of our chunks top-left compared to grids top left.
			       int offset_v, //Same as above but for V
			       int offset_w, //Offset w-plane, generally to mid-point.
			       int wp_min, //Miniumum w-plane
			       int wp_max){ //Maximum w-plane

  int last_wp = wp_min;
  for (int wp = wp_min; wp <= wp_max; wp++){

    

  }


  
}
/******************************
	  Host Functions
*******************************/

//Gets minimum/maximum co-ordinate in a particular baseline.
__host__ inline double lambda_min(struct bl_data *bl_data, double u) {
    return u * (u < 0 ? bl_data->f_max : bl_data->f_min) / c;
}

__host__ inline double lambda_max(struct bl_data *bl_data, double u) {
    return u * (u < 0 ? bl_data->f_min : bl_data->f_max) / c;
}


__host__ inline void init_grid_zero(hipDoubleComplex *uvgrid, int grid_size){

  for(int x = 0; x< grid_size; ++x){
    for(int y = 0; y< grid_size; ++y){
      *(uvgrid+x*grid_size+y) = make_hipDoubleComplex(0.0, 0.0);
    }
  }

}

// Get coarse-grained co-ordinate.
__host__ inline static int coord(int grid_size, double theta,
                 struct bl_data *bl_data,
                 int time, int freq) {
#ifdef ASSUME_UVW_0
    int x = 0, y = 0;
#else
    int x = (int)floor(theta * uvw_lambda(bl_data, time, freq, 0) + .5);
    int y = (int)floor(theta * uvw_lambda(bl_data, time, freq, 1) + .5);
#endif
    return (y+grid_size/2) * grid_size + (x+grid_size/2);
}

// Get coarse-grained co-ordinate.
__host__ inline static int coord_flat(int grid_size, double theta,
                 struct flat_vis_data *vis_data,
                 int vi) {
#ifdef ASSUME_UVW_0
    int x = 0, y = 0;
#else
    int x = (int)floor(theta * vis_data->u[vi] + .5);
    int y = (int)floor(theta * vis_data->v[vi] + .5);
#endif
    return (y+grid_size/2) * grid_size + (x+grid_size/2);
}

// Uniformly weights all visibilities.
__host__ inline void weight(unsigned int *wgrid, int grid_size, double theta,
            struct vis_data *vis) {

  int total_vis=0;
    // Simple uniform weighting
  int bl, time, freq;
    memset(wgrid, 0, grid_size * grid_size * sizeof(unsigned int));
    for (bl = 0; bl < vis->bl_count; bl++) {
        for (time = 0; time < vis->bl[bl].time_count; time++) {
            for (freq = 0; freq < vis->bl[bl].freq_count; freq++) {
                wgrid[coord(grid_size, theta, &vis->bl[bl], time, freq)]++;
		++total_vis;
            }
        }
    }

    
    for (bl = 0; bl < vis->bl_count; bl++) {
        for (time = 0; time < vis->bl[bl].time_count; time++) {
            for (freq = 0; freq < vis->bl[bl].freq_count; freq++) {
                vis->bl[bl].vis[time*vis->bl[bl].freq_count + freq]
                    /= wgrid[coord(grid_size, theta, &vis->bl[bl], time, freq)];
            }
        }
    }

}

// Uniformly weights all visibilities on a flat structre.
__host__ inline void weight_flat(unsigned int *wgrid, int grid_size, double theta,
            struct flat_vis_data *vis) {

    // Simple uniform weighting

    memset(wgrid, 0, grid_size * grid_size * sizeof(unsigned int));
    int vii;

    for (vii = 0; vii<vis->number_of_vis; ++vii){
      wgrid[coord_flat(grid_size, theta, vis, vii)]++;
    }

    for (vii = 0; vii<vis->number_of_vis; ++vii){
      vis->vis[vii] /= wgrid[coord_flat(grid_size, theta, vis, vii)];
    }
      
}


//Shifts middle of image to top left corner, to make sure FFT is correct.
// (Remember to use this again after the FFT too...)
__host__ inline void fft_shift(hipDoubleComplex *uvgrid, int grid_size) {

  // Shift the FFT
  assert(grid_size % 2 == 0);
  int x, y;
  for (y = 0; y < grid_size; y++) {
    for (x = 0; x < grid_size/2; x++) {
      int ix0 = y * grid_size + x;
      int ix1 = (ix0 + (grid_size+1) * (grid_size/2)) % (grid_size*grid_size);
      hipDoubleComplex temp = uvgrid[ix0];
      uvgrid[ix0] = uvgrid[ix1];
      uvgrid[ix1] = temp;
    }
  }
}

//Ensures 2-D array is hermitian symmetric.
__host__ inline void make_hermitian(hipDoubleComplex *uvgrid, int grid_size){

  hipDoubleComplex *p0;

  if (grid_size % 2 == 0) {
    p0 = uvgrid + grid_size + 1;
  }
  else {
    p0 = uvgrid;
  }

  hipDoubleComplex *p1 = uvgrid + grid_size * grid_size - 1;

  while (p0 < p1) {
    hipDoubleComplex g0 = *p0;

    hipCadd(*p0++,hipConj(*p1));
    hipCadd(*p1--,hipConj(g0));
    //    *p0++ += hipConj(*p1);
    //*p1-- += hipConj(g0);
  }

  assert ( p0 == p1 && p0 == uvgrid + (grid_size + 1) * (grid_size/2));
  hipCadd(*p0,hipConj(*p0));
  //  *p0 += hipConj(*p0);


}

//Splits our visibilities up into contiguous bins, for each block to apply.
__host__ inline void bin_flat_visibilities(struct flat_vis_data *vis_bins,
					   struct flat_vis_data *vis,
					   int blocks){

  std::cout << "Binning Visibilities. No. of vis: " << vis->number_of_vis << " No. of Blocks: " << blocks << "\n";
  
  int vis_per_block = vis->number_of_vis / blocks;
  int leftovers = vis->number_of_vis % blocks;


  int i;
  for(i = 0; i < blocks-1; ++i){

    cudaError_check(hipMallocManaged((void**)&(vis_bins+i)->u,
				      sizeof(double) * vis_per_block, hipMemAttachGlobal));
    cudaError_check(hipMallocManaged((void**)&(vis_bins+i)->v,
				      sizeof(double) * vis_per_block, hipMemAttachGlobal));
    cudaError_check(hipMallocManaged((void**)&(vis_bins+i)->w,
				      sizeof(double) * vis_per_block, hipMemAttachGlobal));
    cudaError_check(hipMallocManaged((void**)&(vis_bins+i)->vis,
				      sizeof(double _Complex) * vis_per_block, hipMemAttachGlobal));


    cudaError_check(hipMemcpy((vis_bins+i)->u, vis->u + vis_per_block * i,
			       sizeof(double) * vis_per_block, hipMemcpyDefault));
    cudaError_check(hipMemcpy((vis_bins+i)->v, vis->v + vis_per_block * i,
			       sizeof(double) * vis_per_block, hipMemcpyDefault));
    cudaError_check(hipMemcpy((vis_bins+i)->w, vis->w + vis_per_block * i,
			       sizeof(double) * vis_per_block, hipMemcpyDefault));
    cudaError_check(hipMemcpy((vis_bins+i)->vis, vis->vis + vis_per_block * i,
			       sizeof(double _Complex) * vis_per_block, hipMemcpyDefault));
    (vis_bins+i)->number_of_vis = vis_per_block;
  }
  
  //Last one gets remainders.


  cudaError_check(hipMallocManaged((void**)&(vis_bins+i)->u,
				    sizeof(double) * (vis_per_block + leftovers), hipMemAttachGlobal));
  cudaError_check(hipMallocManaged((void**)&(vis_bins+i)->v,
				    sizeof(double) * (vis_per_block + leftovers), hipMemAttachGlobal));
  cudaError_check(hipMallocManaged((void**)&(vis_bins+i)->w,
				    sizeof(double) * (vis_per_block + leftovers), hipMemAttachGlobal));
  cudaError_check(hipMallocManaged((void**)&(vis_bins+i)->vis,
				    sizeof(double _Complex) * (vis_per_block + leftovers), hipMemAttachGlobal));
    

  
  cudaError_check(hipMemcpy((vis_bins+i)->u, vis->u + vis_per_block * i,
			     sizeof(double) * (vis_per_block+leftovers), hipMemcpyDefault));
  cudaError_check(hipMemcpy((vis_bins+i)->v, vis->v + vis_per_block * i,
			     sizeof(double) * (vis_per_block+leftovers), hipMemcpyDefault));
  cudaError_check(hipMemcpy((vis_bins+i)->w, vis->w + vis_per_block * i,
			     sizeof(double) * (vis_per_block+leftovers), hipMemcpyDefault));
  cudaError_check(hipMemcpy((vis_bins+i)->vis, vis->vis + vis_per_block * i,
			     sizeof(double _Complex) * (vis_per_block+leftovers), hipMemcpyDefault));
   (vis_bins+i)->number_of_vis = vis_per_block + leftovers;

  
}


//Bins visibilities in u/v for w-towers style subgrids.
__host__ inline void bin_visibilities(struct vis_data *vis, struct bl_data ***bins,
				      int chunk_count, int wincrement, double theta,
				      int grid_size, int chunk_size, int *w_min, int *w_max){

  std::cout << "Binning our visibilities in U/V for our chunks..\n";
  // Determine bounds in w
  double vis_w_min = 0, vis_w_max = 0;
  int bl;
  for (bl = 0; bl < vis->bl_count; bl++) {
    double w_min = lambda_min(&vis->bl[bl], vis->bl[bl].w_min);
    double w_max = lambda_max(&vis->bl[bl], vis->bl[bl].w_max);
    if (w_min < vis_w_min) { vis_w_min = w_min; }
    if (w_max > vis_w_max) { vis_w_max = w_max; }
  }

  int wp_min = (int) floor(vis_w_min / wincrement + 0.5);
  int wp_max = (int) floor(vis_w_max / wincrement + 0.5);

  *w_min = wp_min;
  *w_max = wp_max;

  // Bin in uv
  int bins_size = sizeof(void *) * chunk_count * chunk_count;
  //cudaError_check(hipMallocManaged(&bins, bins_size, hipMemAttachGlobal));
  //cudaError_check(hipMemset(bins, 0, bins_size));
  
  int bins_count_size = sizeof(int) * chunk_count * chunk_count;
  int *bins_count;
  cudaError_check(hipMallocManaged(&bins_count, bins_count_size, hipMemAttachGlobal));
  cudaError_check(hipMemset(bins_count, 0, bins_count_size));
  for (bl = 0; bl < vis->bl_count; bl++) {
    
    // Determine bounds (could be more precise, future work...)
    struct bl_data *bl_data = &vis->bl[bl];
    double u_min = lambda_min(bl_data, bl_data->u_min);
    double u_max = lambda_max(bl_data, bl_data->u_max);
    double v_min = lambda_min(bl_data, bl_data->v_min);
    double v_max = lambda_max(bl_data, bl_data->v_max);
    
    // Determine first/last overlapping grid chunks
    int cx0 = (floor(u_min * theta + 0.5) + grid_size/2) / chunk_size;
    int cx1 = (floor(u_max * theta + 0.5) + grid_size/2) / chunk_size;
    int cy0 = (floor(v_min * theta + 0.5) + grid_size/2) / chunk_size;
    int cy1 = (floor(v_max * theta + 0.5) + grid_size/2) / chunk_size;

    int cy, cx;
    for (cy = cy0; cy <= cy1; cy++) {
      for (cx = cx0; cx <= cx1; cx++) {
	// Lazy dynamically sized vector

	
	int bcount = ++bins_count[cy*chunk_count + cx];
	int bcount_p = bcount - 1;

	// This is a horrible way of doing this.
	// Why can't NVIDIA re-implement realloc?
	struct bl_data **bl_data_old = bins[cy*chunk_count + cx];
	cudaError_check(hipMallocManaged(&bins[cy*chunk_count + cx],sizeof(void *) * bcount, hipMemAttachGlobal));
	cudaError_check(hipMemcpy((void **)bins[cy*chunk_count + cx], (void **)bl_data_old,sizeof(void *) * bcount_p, hipMemcpyDefault)); // Think the --bcount is wrong.
	cudaError_check(hipFree((void **)bl_data_old));
	bins[cy*chunk_count + cx][bcount-1] = bl_data;

      }
    }
  }
  std::cout << "Bins processed: " << bins_size << "\n";
}

//W-Towers Wrapper.
__host__ hipError_t wtowers_CUDA(const char* visfile, const char* wkernfile, int grid_size,
			   double theta,  double lambda, double bl_min, double bl_max,
				  int subgrid_size, int subgrid_margin, double wincrement){

  //API Variables
  hipError_t error = (hipError_t)0;
  hipblasStatus_t stat;
  hipblasHandle_t handle;
  //For Benchmarking.
  hipEvent_t start, stop;
  float elapsedTime;

  // Load visibility and w-kernel data from HDF5 files.
  struct vis_data *vis_dat;
  struct w_kernel_data *wkern_dat;

  cudaError_check(hipMallocManaged((void **)&vis_dat, sizeof(struct vis_data), hipMemAttachGlobal));
  cudaError_check(hipMallocManaged((void **)&wkern_dat, sizeof(struct w_kernel_data), hipMemAttachGlobal));

  int error_hdf5;
  error_hdf5 = load_vis_CUDA(visfile,vis_dat,bl_min,1000);
  if (error_hdf5) {
    std::cout << "Failed to Load Visibilities \n";
    return error;
  }
  error_hdf5 = load_wkern_CUDA(wkernfile, theta, wkern_dat);
  if (error_hdf5) {
    std::cout << "Failed to Load W-Kernels \n";
    return error;
  }


  //Allocate our main grid.
  
  int total_gs = grid_size * grid_size;
  
  hipDoubleComplex *grid_dev, *grid_host;
  cudaError_check(hipMalloc((void **)&grid_dev, total_gs * sizeof(hipDoubleComplex)));
  cudaError_check(hipHostMalloc((void **)&grid_host, total_gs * sizeof(hipDoubleComplex)));

  int subgrid_mem_size = sizeof(hipDoubleComplex) * subgrid_size * subgrid_size;


  //Create the fresnel interference pattern for the W-Dimension
  //See Tim Cornwells paper on W-Projection for more information.
  
  hipDoubleComplex *wtransfer;
  cudaError_check(hipMallocManaged((void **)&wtransfer, subgrid_mem_size, hipMemAttachGlobal));

  int x,y;
  for (y=0; y < subgrid_size; ++y){

    for (x=0; x < subgrid_size; ++x){

      double l = theta * (double)(x - subgrid_size / 2) / subgrid_size;
      double m = theta * (double)(y - subgrid_size / 2) / subgrid_size;
      double ph = wincrement * (1 - sqrt(1 - l*l - m*m));

      hipDoubleComplex wtrans = make_hipDoubleComplex(0, 2 * M_PI * ph);
      wtransfer[y * subgrid_size + x] = cu_cexp_d(wtrans);
    }

  }

  //Initialise cublas handle.

  stat = hipblasCreate(&handle);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    std::cout<< "cuBLAS initialisation failed. \n";
    error = (hipError_t)1;
    return error;
  }

  
  //Create FFT Plans for our frequent fft's.

  hipfftHandle fft_plan;
  hipfftPlan2d(&fft_plan,subgrid_size,subgrid_size,HIPFFT_D2Z);


  //Allocate subgrids/subimgs on the GPU
  
  assert( grid_size % subgrid_size == 0);
  int chunk_count_1d = grid_size / subgrid_size;
  int total_chunks = chunk_count_1d * chunk_count_1d;

  hipDoubleComplex **subgrids, **subimgs;

  cudaError_check(hipMallocManaged(&subgrids, total_chunks * sizeof(hipDoubleComplex)));
  cudaError_check(hipMallocManaged(&subimgs, total_chunks * sizeof(hipDoubleComplex)));

  //Create streams for each tower and allocate our chunks in unified memory.
  
  hipStream_t *streams = (hipStream_t *) malloc(total_chunks * sizeof(hipStream_t));
  for(int i = 0; i < total_chunks; ++i){

    hipStreamCreate(&streams[i]);

    cudaError_check(hipMallocManaged(subgrids + i, subgrid_mem_size * sizeof(hipDoubleComplex)));
    cudaError_check(hipMallocManaged(subimgs + i, subgrid_mem_size * sizeof(hipDoubleComplex)));

  }

  struct bl_data ***bins;
  int wp_max, wp_min;
  cudaError_check(hipMallocManaged(&bins, total_chunks * sizeof(void *), hipMemAttachGlobal));
  cudaError_check(hipMemset(bins, 0, total_chunks * sizeof(void *)));
  

  bin_visibilities(vis_dat, bins, chunk_count_1d, wincrement, theta, grid_size, subgrid_size, &wp_max, &wp_min);


  hipEventCreate(&start);
  hipEventRecord(start,0);
  int shift_threads = 64;
  int shift_blocks = subgrid_size / shift_threads;

  dim3 blocks_shift(shift_blocks,shift_blocks);
  
  

  // Lets get gridding!
  for(int chunk = 0; chunk < total_chunks; ++chunk){

    int chunk_x = chunk % chunk_count_1d;
    int chunk_y = chunk / chunk_count_1d;

    int offset_x = chunk_x * subgrid_size;
    int offset_y = chunk_y * subgrid_size;

    scatter_grid_kernel <<< 1, 64, 0, streams[chunk] >>>
			     (bins[chunk], vis_dat->bl_count,
			      vis_dat, wkern_dat, subgrids[chunk], wkern_dat->size_x,
			      subgrid_size, subgrid_size, wkern_dat->w_step, theta,
			      offset_x, offset_y, 0);
    fresnel_blas_mmul(handle, subgrids[chunk], wtransfer, subimgs[chunk], subgrid_size);

			     

  }

  hipEventCreate(&stop);
  hipEventRecord(stop,0);

  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime,start,stop);

  
  
  return error;

}


// Pure W-Projection on a Hierarchical Dataset. (AoS)
__host__ hipError_t wprojection_CUDA(const char* visfile, const char* wkernfile, int grid_size,
				      double theta,  double lambda, double bl_min, double bl_max,
				      int threads_per_block){

  //For Benchmarking.
  
  hipError_t error = (hipError_t)0; //Initialise as CUDA_Success
  hipEvent_t start, stop;
  float elapsedTime;

  // Load visibility and w-kernel data from HDF5 files.
  
  struct vis_data *vis_dat;
  struct w_kernel_data *wkern_dat;

  cudaError_check(hipMallocManaged((void **)&vis_dat, sizeof(struct vis_data), hipMemAttachGlobal));
  cudaError_check(hipMallocManaged((void **)&wkern_dat, sizeof(struct w_kernel_data), hipMemAttachGlobal));

  int error_hdf5;
  error_hdf5 = load_vis_CUDA(visfile,vis_dat,bl_min,bl_max);
  if (error_hdf5) {
    std::cout << "Failed to Load Visibilities \n";
    return error;
  }
  error_hdf5 = load_wkern_CUDA(wkernfile, theta, wkern_dat);
  if (error_hdf5) {
    std::cout << "Failed to Load W-Kernels \n";
    return error;
  }

  

  //Allocate our main grid.
  
  int total_gs = grid_size * grid_size;
  
  hipDoubleComplex *grid_dev, *grid_host;
  cudaError_check(hipMalloc((void **)&grid_dev, total_gs * sizeof(hipDoubleComplex)));
  cudaError_check(hipHostMalloc((void **)&grid_host, total_gs * sizeof(hipDoubleComplex)));


  
  //int blocks = total_gs / 256;

  //Weight visibilities

  weight((unsigned int *)grid_host, grid_size, theta, vis_dat);
    
  struct bl_data **bl_d = &vis_dat->bl;
  
  std::cout << "Inititalise scatter gridder... \n";

  hipEventCreate(&start);
  hipEventRecord(start, 0);

  //scatter_grid_kernel_flat <<< 16, 32 >>> (flat_vis_dat, wkern_dat, grid_dev, wkern_dat->size_x,
  //					  grid_size, grid_size, wkern_dat->w_step, theta, 0, 0, 0);
  //
  scatter_grid_kernel <<< 16 , 32 >>> (bl_d,vis_dat->bl_count,
				       vis_dat, wkern_dat, grid_dev, wkern_dat->size_x,
				       grid_size, grid_size, wkern_dat->w_step, theta, 0, 0, 0);
  hipEventCreate(&stop);
  hipEventRecord(stop, 0);

  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime,start,stop);

  std::cout << "Scatter Gridder Elapsed Time: " << elapsedTime/1000.0 << " seconds\n";


  free_vis_CUDA(vis_dat);
  //Transfer back to host.
  cudaError_check(hipMemcpy(grid_host, grid_dev, total_gs * sizeof(hipDoubleComplex),
			     hipMemcpyDeviceToHost));



//Write Image to disk on host.

  std::ofstream image_pref ("pre_fft.out", std::ofstream::out | std::ofstream::binary);
  std::cout << "Writing Image to File... \n";

  double *row;
  cudaError_check(hipHostMalloc(&row, grid_size * sizeof(double)));

      
  for(int i = 0; i < grid_size; ++i){

    for(int j = 0; j< grid_size; ++j){
      
      row[j] = hipCreal(grid_host[i*grid_size + j]);
    }
    image_pref.write((char*)row, sizeof(double) * grid_size);
  }
  image_pref.close();


  
  fft_shift(grid_host, grid_size);
  make_hermitian(grid_host, grid_size);


  
  cudaError_check(hipMemcpy(grid_dev, grid_host, total_gs * sizeof(hipDoubleComplex),
			     hipMemcpyHostToDevice));
  
  std::cout << "Executing iFFT back to Image Space... \n";
  
  hipfftHandle fft_plan;
  cuFFTError_check(hipfftPlan2d(&fft_plan,grid_size,grid_size,HIPFFT_Z2Z));
  cuFFTError_check(hipfftExecZ2Z(fft_plan, grid_dev, grid_dev, HIPFFT_BACKWARD));

  //Transfer back to host.
  cudaError_check(hipMemcpy(grid_host, grid_dev, total_gs * sizeof(hipDoubleComplex),
			     hipMemcpyDeviceToHost));


  //Write Image to disk on host.

  std::ofstream image_f ("image.out", std::ofstream::out | std::ofstream::binary);
  std::cout << "Writing Image to File... \n";

  //double *row;
  //cudaError_check(hipHostMalloc(&row, grid_size * sizeof(double)));
  fft_shift(grid_host,grid_size);
  for(int i = 0; i < grid_size; ++i){

    for(int j = 0; j< grid_size; ++j){

      row[j] = hipCreal(grid_host[i*grid_size + j]);
    }
    image_f.write((char*)row, sizeof(double) * grid_size);
  }

  image_f.close();

  //Check it actually ran...
  hipError_t err = hipGetLastError();
  

  std::cout << "Error: " << hipGetErrorString(err) << "\n";
  return err;
}

// W-Project on flat SoA Dataset.
__host__ hipError_t wprojection_CUDA_flat(const char* visfile, const char* wkernfile, int grid_size,
				      double theta,  double lambda, double bl_min, double bl_max,
				      int threads_per_block){

  hipDeviceSetCacheConfig(hipFuncCachePreferL1);
  //For Benchmarking.
  
  hipError_t error = (hipError_t)0;
  hipEvent_t start, stop;
  float elapsedTime;

  // Load visibility and w-kernel data from HDF5 files.
  
  struct vis_data *vis_dat = (struct vis_data*)malloc(sizeof(struct vis_data));

  vis_dat->antenna_count = 0;
  
  struct w_kernel_data *wkern_dat;

  //hipHostMalloc((void **)&vis_dat, sizeof(struct vis_data));
  //cudaError_check(hipHostMalloc((void **)&vis_dat, sizeof(struct vis_data)));
  cudaError_check(hipMallocManaged((void **)&wkern_dat, sizeof(struct w_kernel_data), hipMemAttachGlobal));

  int error_hdf5;
  error_hdf5 = load_vis(visfile,vis_dat,bl_min,bl_max);
  if (error_hdf5) {
    std::cout << "Failed to Load Visibilities \n";
    return error;
  }
  error_hdf5 = load_wkern_CUDA(wkernfile, theta, wkern_dat);
  if (error_hdf5) {
    std::cout << "Failed to Load W-Kernels \n";
    return error;
  }

  //Allocate our main grid.
  
  int total_gs = grid_size * grid_size;
  
  hipDoubleComplex *grid_dev, *grid_host;
  cudaError_check(hipMalloc((void **)&grid_dev, total_gs * sizeof(hipDoubleComplex)));
  cudaError_check(hipHostMalloc((void **)&grid_host, total_gs * sizeof(hipDoubleComplex)));

  //Make sure our grids are all zero.
  
  cudaError_check(hipMemset(grid_dev, 0, total_gs * sizeof(hipDoubleComplex)));
  cudaError_check(hipMemset(grid_host, 0, total_gs * sizeof(hipDoubleComplex)));

  
  struct flat_vis_data *flat_vis_dat;
  cudaError_check(hipHostMalloc((void**)&flat_vis_dat, sizeof(struct flat_vis_data)));

  //Flatten the visibilities and weight them.
  flatten_visibilities_CUDA(vis_dat,flat_vis_dat);
  weight_flat((unsigned int *)grid_host, grid_size, theta, flat_vis_dat);


  
  //Now bin them per block.
  struct flat_vis_data *vis_bins;
  cudaError_check(hipMallocManaged((void**)&vis_bins, sizeof(struct flat_vis_data) * 1024, hipMemAttachGlobal));

  bin_flat_visibilities(vis_bins, flat_vis_dat, 1024);
  

  //Get scattering..
  std::cout << "Inititalise scatter gridder... \n";
  hipEventCreate(&start);
  hipEventRecord(start, 0);

  scatter_grid_kernel_flat <<< 1024, 256 >>> (vis_bins, wkern_dat, grid_dev, wkern_dat->size_x,
  					  grid_size, grid_size, wkern_dat->w_step, theta, 0, 0, 0);
  
  hipEventCreate(&stop);
  hipEventRecord(stop, 0);

  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime,start,stop);

  std::cout << "Scatter Gridder Elapsed Time: " << elapsedTime/1000.0 << " seconds\n";

  
  cudaError_check(hipDeviceSynchronize());

  //Shift our grid to the right position for the FFT.
  int fft_gs = 32;
  int fft_bs = grid_size / fft_gs;
  
  dim3 dimBlock(fft_bs,fft_bs);
  dim3 dimGrid(fft_gs,fft_gs);
  fft_shift_kernel <<< dimBlock, dimGrid >>> (grid_dev, grid_size);
  //Transfer back to host.
  cudaError_check(hipMemcpy(grid_host, grid_dev, total_gs * sizeof(hipDoubleComplex),
			     hipMemcpyDeviceToHost));



  //Write Grid to disk on host.

  std::ofstream image_pref ("pre_fft.out", std::ofstream::out | std::ofstream::binary);
  std::cout << "Writing Image to File... \n";
  
  double *row;
  cudaError_check(hipHostMalloc(&row, grid_size * sizeof(double)));

     
  for(int i = 0; i < grid_size; ++i){

    for(int j = 0; j< grid_size; ++j){
      
      row[j] = hipCreal(grid_host[i*grid_size + j]);
    }
    image_pref.write((char*)row, sizeof(double) * grid_size);
  }
  image_pref.close();


  
  
  
  make_hermitian(grid_host, grid_size);


  
  cudaError_check(hipMemcpy(grid_dev, grid_host, total_gs * sizeof(hipDoubleComplex),
  			     hipMemcpyHostToDevice));
  
  std::cout << "Executing iFFT back to Image Space... \n";
  
  hipfftHandle fft_plan;
  cuFFTError_check(hipfftPlan2d(&fft_plan,grid_size,grid_size,HIPFFT_Z2Z));
  cuFFTError_check(hipfftExecZ2Z(fft_plan, grid_dev, grid_dev, HIPFFT_BACKWARD));
  fft_shift_kernel <<< dimBlock, dimGrid >>> (grid_dev, grid_size);
  //Transfer back to host.
  cudaError_check(hipMemcpy(grid_host, grid_dev, total_gs * sizeof(hipDoubleComplex),
			     hipMemcpyDeviceToHost));


  //Write Image to disk on host.

  std::ofstream image_f ("image.out", std::ofstream::out | std::ofstream::binary);
  std::cout << "Writing Image to File... \n";

  //  fft_shift(grid_host,grid_size);
  for(int i = 0; i < grid_size; ++i){

    for(int j = 0; j< grid_size; ++j){

      row[j] = hipCreal(grid_host[i*grid_size + j]);
    }
    image_f.write((char*)row, sizeof(double) * grid_size);
  }

  image_f.close();

  //Check it actually ran...
  hipError_t err = hipGetLastError();
  std::cout << "Error: " << hipGetErrorString(err) << "\n";

  cudaError_check(hipDeviceReset());
  
  return err;
} 
