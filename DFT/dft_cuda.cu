#include "hip/hip_runtime.h"
//C++ Includes
#include <iostream>
#include <cstdlib>

//CUDA Includes
#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>
#include "hip/hip_runtime.h"
#include "math.h"
#include "hip/hip_runtime_api.h"

//Our include
#include "dft_common.h"


//Using unified memory instead of a deep copy.

// This calculates the DFT at a specific point on the grid.
// It adds to a local register and then does an atomic add to device memory.
__device__ hipDoubleComplex calculate_dft_sum(struct vis_data *vis, double l, double m){

  //nvcc should put this in a register.
  hipDoubleComplex grid_point = make_hipDoubleComplex(0.0,0.0);
  
  for (int bl = 0; bl < vis->bl_count; ++bl){

    for (int time = 0; time < vis->bl[bl].time_count; ++time){

      for (int freq = 0; freq < vis->bl[bl].freq_count; ++freq){

	//This step is quite convoluted due to mixing C and CUDA compelx datatypes..
	hipDoubleComplex visibility;
	double __complex__ visibility_c = vis->bl[bl].vis[time*vis->bl[bl].freq_count + freq];
	memcpy(&visibility, &visibility_c, sizeof(double __complex__));
	

	//nvcc should optimise this section.
	double subang1 = m * vis->bl[bl].uvw[time*vis->bl[bl].freq_count + freq];
	double subang2 = l * vis->bl[bl].uvw[time*vis->bl[bl].freq_count + freq + 1];
	double subang3 = (sqrtf(1-l*l-m*m)-1) * vis->bl[bl].uvw[time*vis->bl[bl].freq_count +
								freq + 2];

	double angle = 2 * M_PI * subang1 + subang2 + subang3;

	double real_p = hipCreal(visibility) * cos(angle) + hipCimag(visibility) * sin(angle);
	double complex_p = -hipCreal(visibility) * sin(angle) + hipCimag(visibility) * cos(angle);

	//Add these to our grid_point so far.
	grid_point = hipCadd(grid_point, make_hipDoubleComplex(real_p, complex_p));
							       

      }
    }
  }
  

  return grid_point; //Placeholder
}


//Executes a direct DFT from a given visibility dataset.
__global__ void image_dft(struct vis_data *vis, hipDoubleComplex *uvgrid, int grid_size,
			  double lambda, int iter, int N){

  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  
  int y = floor( (double)(idx / grid_size) ); //Typecast makes sure that we use the CUDA floor, not the C one.
  int x = idx % grid_size;

  double l = (y - grid_size / 2)/lambda;
  double m = (x - grid_size / 2)/lambda;
  
  uvgrid[idx] = calculate_dft_sum(vis, l, m);


}


//This wraps the CUDA Kernel. Otherwise g++ doesn't recognise the <<< operator.
__host__ void image_dft_host(const char* visfile, int grid_size,
		    double theta,  double lambda, double bl_min, double bl_max,
		    int iter){

  hipError_t error;

  hipEvent_t start, stop;
  float elapsedTime;
  
  //  error = hipMallocManaged(reinterpret_cast<void **>(&vis_dat),sizeof(struct vis_data), hipMemAttachGlobal);
  struct vis_data vis_dat;
  int viserr = load_vis(visfile,&vis_dat,bl_min,bl_max);

  if (viserr){
    std::cout << "Failed to Load Visibilities \n";
    return; //Kill Program.
  }

  // Now to get visibilities to the device.

  struct vis_data vis_dat_gpu;

  //Declare our grid.
  //int grid_size = floor(lambda * theta);

  std::cout << "Theta: " << theta << "\n";
  std::cout << "Lambda: " << lambda << "\n";
  std::cout << "Grid Size: " << grid_size << " x " << grid_size << "\n";
  std::cout << "Grid Memory: " << (grid_size * grid_size * sizeof(double _Complex))/1e9 << "\n";


  
  std::cout<<"\n\n Executing Kernel \n";
  int total_gs = grid_size * grid_size;

  std::cout<<"Total Size: " << total_gs << "\n\n";
  

  hipDoubleComplex *grid_dev;
  error = hipMalloc((void **)&grid_dev, grid_size * grid_size * sizeof(hipDoubleComplex));
  if (error == hipSuccess){

    hipEventCreate(&start);
    hipEventRecord(start, 0);
    image_dft <<< 4096,1024>>> (&vis_dat, grid_dev, grid_size, lambda, iter, total_gs);
    hipEventCreate(&stop);
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "Elapsed Time: " << elapsedTime << "\n";
  }
  else {

    std::cout << "Memory Allocation Failed. \n";

  }


  std::cout << "DFT Value: " << hipCreal(grid_dev[500]);
  

  //Check it actually ran...
  hipError_t err = hipGetLastError();

  std::cout << "Error: " << hipGetErrorString(err) << "\n";
  

}