#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>

//CUDA Includes
#include <hip/hip_complex.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

//Our include
#include "dft_common.h"



// This calculates the DFT at a specific point on the grid.
// It adds to a local register and then does an atomic add to device memory.
__device__ hipDoubleComplex calculate_dft_sum(struct vis_data *vis, double l, double m){

  //nvcc should put this in a register.
  hipDoubleComplex grid_point = make_hipDoubleComplex(0.0,0.0);
  
  for (int bl = 0; bl < vis->bl_count; ++bl){

    for (int time = 0; time < vis->bl[bl].time_count; ++time){

      for (int freq = 0; freq < vis->bl[bl].freq_count; ++freq){

	//This step is quite convoluted due to mixing C and CUDA compelx datatypes..
	hipDoubleComplex visibility;
	double __complex__ visibility_c = vis->bl[bl].vis[time*vis->bl[bl].freq_count + freq];
	memcpy(&visibility, &visibility_c, sizeof(double __complex__));
	

	//nvcc should optimise this section.
	double subang1 = m * vis->bl[bl].uvw[time*vis->bl[bl].freq_count + freq];
	double subang2 = l * vis->bl[bl].uvw[time*vis->bl[bl].freq_count + freq + 1];
	double subang3 = (sqrtf(1-l*l-m*m)-1) * vis->bl[bl].uvw[time*vis->bl[bl].freq_count +
								freq + 2];

	double angle = 2 * M_PI * subang1 + subang2 + subang3;

	double real_p = hipCreal(visibility) * cos(angle) + hipCimag(visibility) * sin(angle);
	double complex_p = -hipCreal(visibility) * sin(angle) + hipCimag(visibility) * cos(angle);

	//Add these to our grid_point so far.
	grid_point = hipCadd(grid_point, make_hipDoubleComplex(real_p, complex_p));
							       

      }
    }
  }
  

  return grid_point; //Placeholder
}
