#include "hip/hip_runtime.h"
//C++ Includes
#include <iostream>
#include <fstream>
#include <cstdlib>

//CUDA Includes
#include <hip/hip_complex.h>
#include "hip/hip_runtime.h"
#include "math.h"
#include "hip/hip_runtime_api.h"

//Our include
#include "dft_common.h"


#define cudaError_check(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

//Using unified memory instead of a deep copy.
// This calculates the DFT at a specific point on the grid.
// It adds to a local register and then does an atomic add to device memory.
__device__ hipDoubleComplex calculate_dft_sum(struct vis_data *vis, double l, double m){

  //nvcc should put this in a register.
  hipDoubleComplex grid_point = make_hipDoubleComplex(0.0,0.0);
  
  for (int bl = 0; bl < vis->bl_count; ++bl){

    for (int time = 0; time < vis->bl[bl].time_count; ++time){

      for (int freq = 0; freq < vis->bl[bl].freq_count; ++freq){


	//TODO: Put pointer cast in instead like in WTowers. Haven't got time to test now :(
	//This step is quite convoluted due to mixing C and CUDA complex datatypes..
	hipDoubleComplex visibility;
	double __complex__ visibility_c = vis->bl[bl].vis[time*vis->bl[bl].freq_count + freq];
	memcpy(&visibility, &visibility_c, sizeof(double __complex__));
	

	//nvcc should optimise this section.
	double subang1 = l * vis->bl[bl].uvw[time*vis->bl[bl].freq_count + freq];
	double subang2 = m * vis->bl[bl].uvw[time*vis->bl[bl].freq_count + freq + 1];
	double subang3 = (sqrtf(1-l*l-m*m)-1) * vis->bl[bl].uvw[time*vis->bl[bl].freq_count +
								freq + 2];

	double angle = 2 * M_PI * (subang1 + subang2 + subang3);

	double real_p = hipCreal(visibility) * cos(angle) + hipCimag(visibility) * sin(angle);
	double complex_p = -hipCreal(visibility) * sin(angle) + hipCimag(visibility) * cos(angle);

	//Add these to our grid_point so far.
	grid_point = hipCadd(grid_point, make_hipDoubleComplex(real_p, complex_p));
							       

      }
    }
  }
  

  return grid_point; //Placeholder
}


//Executes a direct DFT from a given visibility dataset.
__global__ void image_dft(struct vis_data *vis, hipDoubleComplex *uvgrid,
			  int grid_size, double lambda){

  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  
  int y = floor( (double)(idx / grid_size) ); //Typecast makes sure that we use the CUDA floor, not the C one.
  int x = idx % grid_size;

  double l = ((y - grid_size / 2)/lambda) * resolution;
  double m = ((x - grid_size / 2)/lambda) * resolution;
  
  uvgrid[idx] = calculate_dft_sum(vis, l, m);


}


//This wraps the CUDA Kernel. Otherwise g++ doesn't recognise the <<< operator.
__host__ hipError_t image_dft_host(const char* visfile, int grid_size,
				    double theta,  double lambda, double bl_min, double bl_max,
				    int blocks, int threads_block){

  hipError_t error = hipSuccess;

  hipEvent_t start, stop;
  float elapsedTime;

  struct vis_data *vis_dat;
  cudaError_check(hipMallocManaged((void **)&vis_dat,sizeof(struct vis_data), hipMemAttachGlobal));

  int viserr = load_vis_CUDA(visfile,vis_dat,bl_min,bl_max);

  if (viserr){
    std::cout << "Failed to Load Visibilities \n";
    return error; //Kill Program.
  }  

  //Declare our grid.
  //int grid_size = floor(lambda * theta);

  std::cout << "Theta: " << theta << "\n";
  std::cout << "Lambda: " << lambda << "\n";
  std::cout << "Grid Size: " << grid_size << " x " << grid_size << "\n";
  std::cout << "Grid Memory: " << (grid_size * grid_size * sizeof(double _Complex))/1e9 << "\n";


  
  std::cout<<"\n\n Executing Kernel \n";
  int total_gs = grid_size * grid_size;

  std::cout<<"Total Size: " << total_gs << "\n\n";
  

  hipDoubleComplex *grid_dev,*grid_host;
  cudaError_check(hipMalloc((void **)&grid_dev, grid_size * grid_size * sizeof(hipDoubleComplex)));
  cudaError_check(hipHostMalloc((void **)&grid_host, grid_size * grid_size * sizeof(hipDoubleComplex)));
  hipEventCreate(&start);
  hipEventRecord(start, 0);
  image_dft <<< blocks , threads_block >>> (vis_dat, grid_dev, grid_size, lambda);
  hipEventCreate(&stop);
  hipEventRecord(stop, 0);

  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);

  std::cout << "Elapsed Time: " << elapsedTime << "\n";
  


  //  std::cout << "DFT Value: " << hipCreal(grid_dev[500]);
  std::cout << "Copying grid from device to host... \n";
  cudaError_check(hipMemcpy(grid_host,grid_dev, grid_size * grid_size * sizeof(hipDoubleComplex),
			     hipMemcpyDeviceToHost));
  //Create Image File


  
  std::ofstream image_f ("image.out", std::ofstream::out | std::ofstream::binary);
  std::cout << "Writing Image to File... \n";
  

  

  
  
  
  //Write Image to disk on host.
  double *row;
  cudaError_check(hipHostMalloc(&row,grid_size * sizeof(double)));
    
  for(int i = 0; i < grid_size; i++){

    for(int j = 0; j< grid_size; j++){

      row[j] = hipCreal(grid_host[i*grid_size + j]);

    }
    image_f.write((char*)row, sizeof(double) * grid_size);
  }

  image_f.close();

  //Check it actually ran...
  hipError_t err = hipGetLastError();

  std::cout << "Error: " << hipGetErrorString(err) << "\n";
  return err;

}
